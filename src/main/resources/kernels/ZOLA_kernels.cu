#include "hip/hip_runtime.h"


//#include "hipblas.h"
//#include "hipblas.h"
 
#include <thrust/sort.h>





extern "C"
__global__ void vec_test1(int n,float *d_A,int size) {
	
    //float sum = thrust::reduce(thrust::seq, d_A, d_A + size);
	
	//thrust::sort_by_key(thrust::device,d_A, d_A + size, index);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id ==0)
    {
		thrust::sort(thrust::seq,d_A, d_A + size);
		printf("max side result = %f     %d\n", *(d_A+size-1),n);
	}

    

}




extern "C"
__global__ void vec_initIndex(int n, int *index)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        index[id] = id;
    }
}






extern "C"
__global__ void vec_computeLocalMaxima(int n, int *localMaxima,int sizeImage,float *input,int sizeFilt,int edgeSizeReject)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
		localMaxima[id]=0;
		int sizeSquare=sizeImage*sizeImage;
		int z=id/sizeSquare;
		int p=id%sizeSquare;
		int x=p/sizeImage;
		int y=p%sizeImage;
		int sizeReject=max(sizeFilt,edgeSizeReject);
		if ((x-sizeReject>=0)&&(y-sizeReject>=0)&&(x+sizeReject<sizeImage)&&(y+sizeReject<sizeImage)){//not edges
			int a,aa,neighbor;
			int ok=1;
			for (a=-sizeFilt;a<=sizeFilt;a++){
				for (a=-sizeFilt;a<=sizeFilt;a++){
					neighbor=z*sizeSquare+(x+a)*sizeImage+(y+aa);
					if (input[id]<input[neighbor]){
						ok=0;
						goto next;
					}
						
				}
			}
			next:
			if (ok==1){
				localMaxima[id]=1;
			}
		}
		
    }
}





extern "C"
__global__ void vec_eraseNonLocalMaxima(int n, float *input,int *localMaxima)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
		if (localMaxima[id]==0){
			input[id]=-1;
		}
		
    }
}





extern "C"
__global__ void vec_sortRows(int n,float *d_A,int * index,int size) {
	
    //float sum = thrust::reduce(thrust::seq, d_A, d_A + size);
	
	//thrust::sort_by_key(thrust::device,d_A, d_A + size, index);
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id ==0)
    {
		thrust::stable_sort_by_key(thrust::seq,d_A, d_A + size, index,thrust::greater<float>());

	}
}




extern "C"
__global__ void vec_set (int n, double *result, double  value)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = value;
    }
}


//=== Vector arithmetic ======================================================

extern "C"
__global__ void vec_add (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] + y[id];
    }
}


extern "C"
__global__ void vec_sub (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] - y[id];
    }
}


extern "C"
__global__ void vec_mul (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] * y[id];
    }
}



extern "C"
__global__ void vec_mul_fl (int n, float *result, float  *x, float  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] * y[id];
    }
}



extern "C"
__global__ void vec_div (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] / y[id];
    }
}

extern "C"
__global__ void vec_negate (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = -x[id];
    }
}




//=== Vector-and-scalar arithmetic ===========================================

extern "C"
__global__ void vec_addScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] + y;
    }
}


extern "C"
__global__ void vec_subScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] - y;
    }
}


extern "C"
__global__ void vec_mulScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] * y;
    }
}


extern "C"
__global__ void vec_divScalar (int n, double *result, double  *x, double  y)
{

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] / y;
    }
}




extern "C"
__global__ void vec_scalarAdd (int n, double *result, double  x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x + y[id];
    }
}


extern "C"
__global__ void vec_scalarSub (int n, double *result, double  x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x - y[id];
    }
}


extern "C"
__global__ void vec_scalarMul (int n, double *result, double  x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x * y[id];
    }
}


extern "C"
__global__ void vec_scalarDiv (int n, double *result, double  x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x / y[id];
    }
}









//=== Vector comparison ======================================================

extern "C"
__global__ void vec_lt (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] < y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_lte (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] <= y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_eq (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] == y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gte (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] >= y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gt (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] > y[id])?1.0:0.0;
    }
}



extern "C"
__global__ void vec_ne (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] != y[id])?1.0:0.0;
    }
}




//=== Vector-and-scalar comparison ===========================================

extern "C"
__global__ void vec_ltScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] < y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_lteScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] <= y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_eqScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] == y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gteScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] >= y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gtScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] > y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_neScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] != y)?1.0:0.0;
    }
}











//=== Vector math (one argument) =============================================


// Calculate the arc cosine of the input argument.
extern "C"
__global__ void vec_acos (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = acos(x[id]);
    }
}


// Calculate the nonnegative arc hyperbolic cosine of the input argument.
extern "C"
__global__ void vec_acosh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = acosh(x[id]);
    }
}


// Calculate the arc sine of the input argument.
extern "C"
__global__ void vec_asin (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = asin(x[id]);
    }
}


// Calculate the arc hyperbolic sine of the input argument.
extern "C"
__global__ void vec_asinh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = asinh(x[id]);
    }
}


// Calculate the arc tangent of the input argument.
extern "C"
__global__ void vec_atan (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = atan(x[id]);
    }
}


// Calculate the arc hyperbolic tangent of the input argument.
extern "C"
__global__ void vec_atanh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = atanh(x[id]);
    }
}


// Calculate the cube root of the input argument.
extern "C"
__global__ void vec_cbrt (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = cbrt(x[id]);
    }
}


// Calculate ceiling of the input argument.
extern "C"
__global__ void vec_ceil (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = ceil(x[id]);
    }
}


// Calculate the cosine of the input argument.
extern "C"
__global__ void vec_cos (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = cos(x[id]);
    }
}


// Calculate the hyperbolic cosine of the input argument.
extern "C"
__global__ void vec_cosh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = cosh(x[id]);
    }
}


// Calculate the cosine of the input argument × p .
extern "C"
__global__ void vec_cospi (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = cospi(x[id]);
    }
}


// Calculate the complementary error function of the input argument.
extern "C"
__global__ void vec_erfc (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = erfc(x[id]);
    }
}


// Calculate the inverse complementary error function of the input argument.
extern "C"
__global__ void vec_erfcinv (int n, double *result, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = erfcinv(y[id]);
    }
}


// Calculate the scaled complementary error function of the input argument.
extern "C"
__global__ void vec_erfcx (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = erfcx(x[id]);
    }
}


// Calculate the error function of the input argument.
extern "C"
__global__ void vec_erf (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = erf(x[id]);
    }
}


// Calculate the inverse error function of the input argument.
extern "C"
__global__ void vec_erfinv (int n, double *result, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = erfinv(y[id]);
    }
}


// Calculate the base 10 exponential of the input argument.
extern "C"
__global__ void vec_exp10 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = exp10(x[id]);
    }
}


// Calculate the base 2 exponential of the input argument.
extern "C"
__global__ void vec_exp2 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = exp2(x[id]);
    }
}


// Calculate the base e exponential of the input argument.
extern "C"
__global__ void vec_exp (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = exp(x[id]);
    }
}


// Calculate the base e exponential of the input argument, minus 1.
extern "C"
__global__ void vec_expm1 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = expm1(x[id]);
    }
}


// Calculate the absolute value of its argument.
extern "C"
__global__ void vec_fabs (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fabs(x[id]);
    }
}


// Calculate the largest integer less than or equal to x.
extern "C"
__global__ void vec_floor (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = floor(x[id]);
    }
}


// Calculate the value of the Bessel function of the first kind of order 0 for the input argument.
extern "C"
__global__ void vec_j0 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = j0(x[id]);
    }
}


// Calculate the value of the Bessel function of the first kind of order 1 for the input argument.
extern "C"
__global__ void vec_j1 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = j1(x[id]);
    }
}


// Calculate the natural logarithm of the absolute value of the gamma function of the input argument.
extern "C"
__global__ void vec_lgamma (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = lgamma(x[id]);
    }
}


// Calculate the base 10 logarithm of the input argument.
extern "C"
__global__ void vec_log10 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = log10(x[id]);
    }
}


// Calculate the value of l o g e ( 1 + x ) .
extern "C"
__global__ void vec_log1p (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = log1p(x[id]);
    }
}


// Calculate the base 2 logarithm of the input argument.
extern "C"
__global__ void vec_log2 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = log2(x[id]);
    }
}


// Calculate the doubleing point representation of the exponent of the input argument.
extern "C"
__global__ void vec_logb (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = logb(x[id]);
    }
}


// Calculate the natural logarithm of the input argument.
extern "C"
__global__ void vec_log (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = log(x[id]);
    }
}


// Calculate the standard normal cumulative distribution function.
extern "C"
__global__ void vec_normcdf (int n, double *result, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = normcdf(y[id]);
    }
}


// Calculate the inverse of the standard normal cumulative distribution function.
extern "C"
__global__ void vec_normcdfinv (int n, double *result, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = normcdfinv(y[id]);
    }
}


// Calculate reciprocal cube root function.
extern "C"
__global__ void vec_rcbrt (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = rcbrt(x[id]);
    }
}


// Round input to nearest integer value in doubleing-point.
extern "C"
__global__ void vec_rint (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = rint(x[id]);
    }
}


// Round to nearest integer value in doubleing-point.
extern "C"
__global__ void vec_round (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = round(x[id]);
    }
}


// Calculate the reciprocal of the square root of the input argument.
extern "C"
__global__ void vec_rsqrt (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = rsqrt(x[id]);
    }
}


// Calculate the sine of the input argument.
extern "C"
__global__ void vec_sin (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = sin(x[id]);
    }
}


// Calculate the hyperbolic sine of the input argument.
extern "C"
__global__ void vec_sinh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = sinh(x[id]);
    }
}


// Calculate the sine of the input argument × p .
extern "C"
__global__ void vec_sinpi (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = sinpi(x[id]);
    }
}


// Calculate the square root of the input argument.
extern "C"
__global__ void vec_sqrt (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = sqrt(x[id]);
    }
}


// Calculate the tangent of the input argument.
extern "C"
__global__ void vec_tan (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = tan(x[id]);
    }
}


// Calculate the hyperbolic tangent of the input argument.
extern "C"
__global__ void vec_tanh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = tanh(x[id]);
    }
}


// Calculate the gamma function of the input argument.
extern "C"
__global__ void vec_tgamma (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = tgamma(x[id]);
    }
}


// Truncate input argument to the integral part.
extern "C"
__global__ void vec_trunc (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = trunc(x[id]);
    }
}


// Calculate the value of the Bessel function of the second kind of order 0 for the input argument.
extern "C"
__global__ void vec_y0 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = y0(x[id]);
    }
}


// Calculate the value of the Bessel function of the second kind of order 1 for the input argument.
extern "C"
__global__ void vec_y1 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = y1(x[id]);
    }
}











//=== Vector math (two arguments) ============================================





// Create value with given magnitude, copying sign of second value.
extern "C"
__global__ void vec_copysign (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = copysign(x[id], y[id]);
    }
}

// Compute the positive difference between x and y.
extern "C"
__global__ void vec_fdim (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fdim(x[id], y[id]);
    }
}

// Divide two doubleing point values.
extern "C"
__global__ void vec_fdivide (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fdivide(x[id], y[id]);
    }
}

// Determine the maximum numeric value of the arguments.
extern "C"
__global__ void vec_fmax (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fmax(x[id], y[id]);
    }
}

// Determine the minimum numeric value of the arguments.
extern "C"
__global__ void vec_fmin (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fmin(x[id], y[id]);
    }
}

// Calculate the doubleing-point remainder of x / y.
extern "C"
__global__ void vec_fmod (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fmod(x[id], y[id]);
    }
}

// Calculate the square root of the sum of squares of two arguments.
extern "C"
__global__ void vec_hypot (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = hypot(x[id], y[id]);
    }
}

// Return next representable single-precision doubleing-point value afer argument.
extern "C"
__global__ void vec_nextafter (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = nextafter(x[id], y[id]);
    }
}

// Calculate the value of first argument to the power of second argument.
extern "C"
__global__ void vec_pow (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = pow(x[id], y[id]);
    }
}

// Compute single-precision doubleing-point remainder.
extern "C"
__global__ void vec_remainder (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = remainder(x[id], y[id]);
    }
}















extern "C"
__global__ void vec_testkernel (int n, double *result, double  *x, double  *y)
{
    
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	for (int j=0;j<100;j++){//stupid...just to test speed
	    result[id] = x[id] * y[id]+sqrt(pow(3.14159,id));;;
	}



}









extern "C"
__global__ void vec_computePSF_phase (int n, double *realOutput, double *imagOutput, double  *kx, double  *ky, double  *kz, double  *pupil, double  *phase,double dx, double dy, double dz)
{
    double x;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        x= kx[id]*dx + ky[id]*dy + kz[id]*dz + phase[id];
		realOutput[id]=pupil[id]*cos(x);
		imagOutput[id]=pupil[id]*sin(x);
		//realOutput[id]=phase[id];
		//imagOutput[id]=sin(phase[id]);
    }



}



extern "C"
__global__ void vec_computePSF_phaseN (int n, double  *kx, double  *ky, double  *kz, double  *pupil, double  *phase,double dx, double dy, double dz, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, double *fft)
{
    double x;

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        x= kx[id]*dx + ky[id]*dy + kz[id]*dz + phase[id];
		
		fft[sparseIndexEvenDisk[id]]=pupil[id]*cos(x);
		fft[sparseIndexOddDisk[id]]=pupil[id]*sin(x);


		
    }



}





extern "C"
__global__ void vec_computePSF_phaseNwithOil (int n, double  *kx, double  *ky, double  *kz,double  *kz_is_imag, double  *kz_oil,double  *kz_oil_is_imag, double  *pupil, double  *phase,double dx, double dy, double dz, double dz_oil, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, double *fft)
{
    double x,y,z;

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		z= kx[id]*dx + ky[id]*dy + phase[id];
        x= z + kz[id]*dz - kz_oil[id]*dz_oil;
		y= z + kz[id]*dz*kz_is_imag[id] - kz_oil[id]*dz_oil*kz_oil_is_imag[id];
		fft[sparseIndexEvenDisk[id]]=pupil[id]*cos(x);
		fft[sparseIndexOddDisk[id]]=pupil[id]*sin(y);


		
    }



}




extern "C"         
__global__ void vec_computePSF_phaseNMany (int n,int sizePart,int sizeTot, double  *kx, double  *ky, double  *kz, double  *pupil, double  *phase,double* position, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, double *fft,int many)
{
    double x;
	
	int u,p;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
			u=id/sizePart;
			p=id%sizePart;
		    x= kx[p]*position[u] + ky[p]*position[u+many] + kz[p]*position[u+2*many] + phase[p];
			fft[sparseIndexEvenDisk[p]+u*sizeTot]=pupil[p]*cos(x);
			fft[sparseIndexOddDisk[p]+u*sizeTot]=pupil[p]*sin(x);
			
			

    }



}




extern "C"         
__global__ void vec_computePSF_phaseNMany_f (int n,int sizePart,int sizeTot, float  *kx, float  *ky, float  *kz, float  *pupil, float  *phase,float* position, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, float *fft,int many)
{
    float x;
	
	int u,p;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
			u=id/sizePart;
			p=id%sizePart;
		    x= kx[p]*position[u] + ky[p]*position[u+many] + kz[p]*position[u+2*many] + phase[p];
			fft[sparseIndexEvenDisk[p]+u*sizeTot]=pupil[p]*cos(x);
			fft[sparseIndexOddDisk[p]+u*sizeTot]=pupil[p]*sin(x);
			
			

    }



}




extern "C"         
__global__ void vec_computePSF_phaseNManywithOil_f (int n,int sizePart,int sizeTot, float  *kx, float  *ky, float  *kz,float  *kz_is_imag, float  *kz_oil,float  *kz_oil_is_imag, float  *pupil, float  *phase,float* position, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, float *fft,int many)
{
    float x,y,z;
	//float x;
	
	int u,p;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
			u=id/sizePart;
			p=id%sizePart;

			//x= kx[p]*position[u] + ky[p]*position[u+many] + phase[p] + kz[p]*position[u+2*many]*kz_is_imag[p] - kz_oil[p]*position[u+3*many]*kz_oil_is_imag[p];
			//fft[sparseIndexEvenDisk[p]+u*sizeTot]=pupil[p]*cos(x);
			//fft[sparseIndexOddDisk[p]+u*sizeTot]=pupil[p]*sin(x);
			
			
			z= kx[p]*position[u] + ky[p]*position[u+many] + phase[p];
		    x= z + kz[p]*position[u+2*many] - kz_oil[p]*position[u+3*many];
			y= z + kz[p]*position[u+2*many]*kz_is_imag[p] - kz_oil[p]*position[u+3*many]*kz_oil_is_imag[p];
			fft[sparseIndexEvenDisk[p]+u*sizeTot]=pupil[p]*cos(x);
			fft[sparseIndexOddDisk[p]+u*sizeTot]=pupil[p]*sin(y);
			
			

    }



}



extern "C"         
__global__ void vec_thetest(int n,float  *kz_is_imag)
{
    
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
			
printf("kz -> %d  %f  \n",id,kz_is_imag[id]);
			
			

    }



}



//WARNING : device_sum size should be gridDim.x
__device__ void sumTheBlocks (double *device_sum)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < 1)//lose few time... i agree
    {
		for (int i=1;i<gridDim.x;i++){
			device_sum[0]+=device_sum[i];
		}
		
	}
}


__device__ int truc;
__device__ int barrier;
__device__ int barrier2;
__device__ void initSynchroBlocks(){
	if (threadIdx.x + blockIdx.x * blockDim.x==0){
		barrier=gridDim.x;
		barrier2=gridDim.x;
	}
}

__device__ void synchroBlocks(){
	__syncthreads();
	if (threadIdx.x==0){
		atomicSub( &barrier , 1 );
	}
	//atomicAdd( &truc , 1 );
	//if ( threadIdx.x == 0 )
        //while ( atomicCAS( &barrier , 0 , 0 ) != 0 );//does not work: infinite loop...

    __syncthreads();
}
__device__ void synchroBlocks2(){
	__syncthreads();
	if (threadIdx.x==0){
		atomicSub( &barrier2 , 1 );
	}
	//atomicAdd( &truc , 1 );
	//if ( threadIdx.x == 0 )
        //while ( atomicCAS( &barrier2 , 0 , 0 ) != 0 );//does not work: infinite loop...

    __syncthreads();
}








extern "C"
__device__ void vec_dense2Sparse (int n, double *device_input, int *device_sparse, double *device_output)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		device_output[id]=device_input[device_sparse[id]];
    }
}


extern "C"
__device__ void vec_sparse2Dense (int n, double *device_input, int *device_sparse, double *device_output)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		device_output[device_sparse[id]]=device_input[id];
    }
}



extern "C"
__global__ void vec_computePSF_signal (int n, double *result, double *real, double  *imag,double divide)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        x=real[id]/divide;
		y=imag[id]/divide;
		result[id]=x*x+y*y;
    }



}




extern "C"
__global__ void vec_computePSF_signalN (int n, double *result, double divide, int *sparseIndexEvenShiftOutput, int *sparseIndexOddShiftOutput, double *fft)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		x=fft[sparseIndexEvenShiftOutput[id]]/divide;
		y=fft[sparseIndexOddShiftOutput[id]]/divide;
		result[id]=x*x+y*y;
    }



}


extern "C"
__global__ void vec_computePSF_signalN2 (int n, double divide, int *sparseIndexEvenShiftOutput, int *sparseIndexOddShiftOutput, double *fft,int *sparseIndexEvenShiftOutputNext,int *sparseIndexOddShiftOutputNext, double *psffft)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		x=fft[sparseIndexEvenShiftOutput[id]]/divide;
		y=fft[sparseIndexOddShiftOutput[id]]/divide;
		psffft[sparseIndexEvenShiftOutputNext[id]]=x*x+y*y;
		psffft[sparseIndexOddShiftOutputNext[id]]=0;

		//psffft[id]=x*x+y*y;
    }



}





extern "C"
__global__ void vec_computePSF_signalN2Many (int n,int sizePart,int sizeTot, double divide, int *sparseIndexEvenShiftOutput, int *sparseIndexOddShiftOutput, double *fft,int *sparseIndexEvenShiftOutputNext,int *sparseIndexOddShiftOutputNext, double *psffft)
{
	int u,p;
	
	
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		u=id/sizePart;
		p=id%sizePart;
		x=fft[sparseIndexEvenShiftOutput[p]+sizeTot*u]/divide;
		y=fft[sparseIndexOddShiftOutput[p]+sizeTot*u]/divide;
		psffft[sparseIndexEvenShiftOutputNext[p]+2*sizePart*u]=x*x+y*y;
		psffft[sparseIndexOddShiftOutputNext[p]+2*sizePart*u]=0;

		//psffft[id]=x*x+y*y;
    }



}






extern "C"
__global__ void vec_computePSF_signalN2Many_f (int n,int sizePart,int sizeTot, float divide, int *sparseIndexEvenShiftOutput, int *sparseIndexOddShiftOutput, float *fft,int *sparseIndexEvenShiftOutputNext,int *sparseIndexOddShiftOutputNext, float *psffft)
{
	int u,p;
	
	
    float x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		u=id/sizePart;
		p=id%sizePart;
		x=fft[sparseIndexEvenShiftOutput[p]+sizeTot*u]/divide;
		y=fft[sparseIndexOddShiftOutput[p]+sizeTot*u]/divide;
		psffft[sparseIndexEvenShiftOutputNext[p]+2*sizePart*u]=x*x+y*y;
		psffft[sparseIndexOddShiftOutputNext[p]+2*sizePart*u]=0;

		//psffft[id]=x*x+y*y;
    }



}




extern "C"
__global__ void vec_computePSF_signalsqrt (int n, double *result, double *real, double  *imag,double divide)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        x=real[id]/divide;
		y=imag[id]/divide;
		result[id]=sqrt(x*x+y*y);
    }



}





extern "C"
__global__ void vec_computePSF_signalNsqrt (int n, double *result, double *fft,double divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        x=fft[sparseIndexEvenShift2D[id]]/divide;
		y=fft[sparseIndexOddShift2D[id]]/divide;
		result[id]=sqrt(x*x+y*y);
    }



}




extern "C"
__global__ void vec_computePSF_signalNsqrtMany (int n, int sizeSubImage,double *result, double *fft,double divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
 	int id2=id%sizeSubImage;
	int id3=id/sizeSubImage;
	if (id < n)
    {
        x=fft[sparseIndexEvenShift2D[id2]+id3*sizeSubImage*2]/divide;
		y=fft[sparseIndexOddShift2D[id2]+id3*sizeSubImage*2]/divide;
		result[id]=sqrt(x*x+y*y);
    }



}




extern "C"
__global__ void vec_computePSF_signalNsqrtMany_f (int n, int sizeSubImage,float *result, float *fft,float divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D)
{
    float x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
 	int id2=id%sizeSubImage;
	int id3=id/sizeSubImage;
	if (id < n)
    {
        x=fft[sparseIndexEvenShift2D[id2]+id3*sizeSubImage*2]/divide;
		y=fft[sparseIndexOddShift2D[id2]+id3*sizeSubImage*2]/divide;
		result[id]=sqrt(x*x+y*y);
    }



}




extern "C"
__global__ void vec_computePSF_signalNsqrtMany_fcrop (int n, int sizeSubImage, int sizeSubImageFull,float *result, float *fft,float divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D)
{
    float x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
 	int id2=id%sizeSubImage;
	int id3=id/sizeSubImage;
	int id4=id3*sizeSubImageFull;
	if (id < n)
    {
        x=fft[sparseIndexEvenShift2D[id2]+id4*2]/divide;
		y=fft[sparseIndexOddShift2D[id2]+id4*2]/divide;
		result[id]=sqrt(x*x+y*y);
    }



}

__device__ void divideBySum (int n, double *result, double *tmpsum)
{
    
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	synchroBlocks2();
	if (id < n)
    {	
		result[id]/=tmpsum[0];
	}

}







extern "C"
__global__ void vec_mulMany (int n, int sizeKernel, double *result, double  *x, double  *kernel)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id%sizeKernel;
    if (id < n)
    {
        result[id] = x[id] * kernel[id2];
    }
}



extern "C"
__global__ void vec_divScalarMany (int n,  int sizeSubImage,double *result, double  *x, double  *div)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeSubImage;
    if (id < n)
    {
		if (div[id2]!=0)
        	result[id] = x[id] / div[id2];
    }
}




extern "C"
__global__ void vec_mulMany_f (int n, int sizeKernel, float *result, float  *x, float  *kernel)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id%sizeKernel;
    if (id < n)
    {
        result[id] = x[id] * kernel[id2];
    }
}



extern "C"
__global__ void vec_computeModelMany1 (int n, int sizeImage, double *result, double  *x, double  *amplitude,double background)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
    if (id < n)
    {
        result[id] = x[id] * amplitude[id2] + background;
    }
}


extern "C"
__global__ void vec_computeModelMany2 (int n, int sizeImage, double *result, double  *x, double  *amplitude,double *background)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
    if (id < n)
    {
        result[id] = x[id] * amplitude[id2] + background[id2];
    }
}



//here, background is 2D
extern "C"
__global__ void vec_computeModelMany3 (int n, int sizeImage, double *result, double  *x, double  *amplitude,double *background)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
	int id3=id%sizeImage;
    if (id < n)
    {

        result[id] = x[id] * amplitude[id2] + background[id3];
		
    }
}



extern "C"
__global__ void vec_computeModelMany1_scmos (int n, int sizeImage, double *result, double  *x, double  *amplitude,double background,double  *scmos)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
	int id3=id%sizeImage;
    if (id < n)
    {
        result[id] = x[id] * amplitude[id2] + background + scmos[id3];
    }
}


extern "C"
__global__ void vec_computeModelMany2_scmos (int n, int sizeImage, double *result, double  *x, double  *amplitude,double *background,double  *scmos)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
	int id3=id%sizeImage;
    if (id < n)
    {
        result[id] = x[id] * amplitude[id2] + background[id2] + scmos[id3];
    }
}



//here, background is 2D
extern "C"
__global__ void vec_computeModelMany3_scmos (int n, int sizeImage, double *result, double  *x, double  *amplitude,double *background,double  *scmos)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
	int id3=id%sizeImage;
    if (id < n)
    {

        result[id] = x[id] * amplitude[id2] + background[id3] + scmos[id3];
		
    }
}



extern "C"
__global__ void vec_divScalarMany_f (int n,  int sizeSubImage,float *result,double *resultDouble, float  *x, float  *div)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeSubImage;
    if (id < n)
    {
		if (div[id2]!=0){
        	result[id] = x[id] / div[id2];
			resultDouble[id] =0;
        	resultDouble[id] =(double)(result[id]);
		}
    }
}








extern "C"
__global__ void vec_computePoissonLikelihood (int n, double *result, double *image, double  *model)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		if (model[id]>0)
			result[id]=model[id]-image[id]*log(model[id]);
		else
			result[id]=10000000;
			
    }



}






//reshuffle: 
//exemple 4 PSF to merge in 2 model
//->>> PSF=1,2,3 merged with PSF=4,5,6
//->>> PSF=7,8,9 merged with PSF=10,11,12
//we need to reshuffle here to use then Dgemv for matrix operation
//1,2,3,4,5,6,7,8,9,10,11,12 -> 1,2,3,7,8,9,4,5,6,10,11,12
extern "C"
__global__ void vec_addPhotonsAndBackgroundManyReshuffle (int n, int sizeSubImage,int numberPSFperModel,double *output, double *input, double *photonAndBackground)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;

	if (id < n)
	{
		int idPSF=id/sizeSubImage;
		int idModel=idPSF/numberPSFperModel;
		int idoffset=id%sizeSubImage;
		int idposit=idPSF%numberPSFperModel;
		int idreshuffle=idModel*sizeSubImage +idposit*sizeSubImage*(n/(sizeSubImage*numberPSFperModel))       +idoffset;
		output[idreshuffle]=input[id]*photonAndBackground[idPSF*2]+photonAndBackground[idPSF*2+1];

	}

}






extern "C"
__global__ void vec_addPhotonsAndBackgroundManyReshuffle_scmos (int n, int sizeSubImage,int numberPSFperModel,double *output, double *input, double *photonAndBackground, double * scmos)
{


//print("to do as previous function");



	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	
	if (id < n)
    {
	int idPSF=id/sizeSubImage;
	int idModel=idPSF/numberPSFperModel;
	int idoffset=id%sizeSubImage;
	int idposit=idPSF%numberPSFperModel;
	int idreshuffle=idModel*sizeSubImage +idposit*sizeSubImage*(n/(sizeSubImage*numberPSFperModel))       +idoffset;
	output[idreshuffle]=input[id]*photonAndBackground[idPSF*2]+photonAndBackground[idPSF*2+1]+scmos[id];

    }

}








extern "C"
__global__ void vec_addPhotonsAndBackgroundMany (int n, int sizeSubImage,double *output, double *input, double *photonAndBackground)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeSubImage;
	if (id < n)
    {
		output[id]=input[id]*photonAndBackground[id2*2]+photonAndBackground[id2*2+1];

    }

}






extern "C"
__global__ void vec_addPhotonsAndBackgroundMany_scmos (int n, int sizeSubImage,double *output, double *input, double *photonAndBackground, double * scmos)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeSubImage;
	if (id < n)
    {
		output[id]=input[id]*photonAndBackground[id2*2]+photonAndBackground[id2*2+1]+scmos[id];

    }

}







extern "C"
__global__ void vec_addPhotonsAndBackgroundMany_f (int n, int sizeSubImage,float *output, float *input, float *photonAndBackground)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeSubImage;
	if (id < n)
    {
		output[id]=input[id]*photonAndBackground[id2*2]+photonAndBackground[id2*2+1];

    }

}






extern "C"
__global__ void vec_double2float (int n, float *output, double *input)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		output[id]=(float)input[id];

    }

}


extern "C"
__global__ void vec_float2double (int n, double *output, float *input)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		output[id]=(double)input[id];

    }

}




//multi kernel complexe conjugate
//*2 because real and imag parts
//n is total size (complex) of kernel divided by 2
//sizeInput is total size (complex) of image divided by 2
extern "C"
__global__ void vec_complexeConjugateKernel (int n,  int sizeInput, float *output, float *input, float *inputKernel)
{
	//n size 
	//int id = 2*(threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = 2*(idy * gridDim.x * blockDim.x + idx);
	int id2=id%(sizeInput*2);
	float real;
	float imag;
	float tmp;
	if (id < n*2)
    {
		real=input[id2]/sqrt((float)sizeInput);
		imag=input[id2+1]/sqrt((float)sizeInput);
		//id : real
		//id+1 : imaginary
		tmp=imag*inputKernel[id+1]+real*inputKernel[id];
		output[id+1]=imag*inputKernel[id]-real*inputKernel[id+1];
		output[id]=tmp;
		

    }

}


//multi kernel correlation result
//n is total size (complex) of kernel divided by 2
//sizeInput is total size (complex) of image divided by 2
extern "C"
__global__ void vec_makeResultCorrelation(int n, int sizeInput, int sizeFullPadded,float *output, float *input, int *sparse)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=(id)%(sizeInput);
	int id3=(id)/(sizeInput);
	if (id < n)
    {
		output[id]=input[sparse[id2]+sizeFullPadded*id3]/sqrt((float)sizeFullPadded/2.);

    }

}





extern "C"
__global__ void vec_divScalarFloat ( int n, float *result, float  *x, float  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = idy * gridDim.x * blockDim.x + idx;

	//int offset = threadIdx.x + blockIdx.x * blockDim.x;
    if (offset < n)
    {
        result[offset] = x[offset] / y;
    }
}





//11 images as input
//25 images as output
//n=3sizesubimages
extern "C"
__global__ void vec_computeCRLB (int n,int sizeMatrix,double *output, double *input,double h)
{
	
	int sizeSubImage=n/(sizeMatrix*sizeMatrix);
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		int p=(id/sizeSubImage);//p=0..24
		int positpix=id%sizeSubImage;//0..sizeSubImage-1
		int line=p/(sizeMatrix);//0..4
		int column=p%(sizeMatrix);//0..4
		double d1=(input[positpix+sizeSubImage*(line*2+2)]-input[positpix+sizeSubImage*(line*2+1)])/(2*h);
		double d2=(input[positpix+sizeSubImage*(column*2+2)]-input[positpix+sizeSubImage*(column*2+1)])/(2*h);



		if (input[positpix]>0){
			output[id]=(d1*d2)/(input[positpix]);
		}
		else{
			output[id]=100000000;
		}
		

    }

}







extern "C"
__global__ void vec_divCorrelation (int n, float  *x,int sizeImage, float  *varImage,float  *varPSF)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
		int lengthImage=sizeImage*sizeImage;
		int positInImage=id%lengthImage;
		int zposit=id/lengthImage;
		float div=(varImage[positInImage]*varPSF[zposit]);
		if (div>0){
			x[id] = x[id] / sqrt(div);
		}
        else{
			x[id] = -1;
		}
    }
}





/*
#include <stdio.h>

int main() {

	//vec_initFFT<<<1,1>>>(16);

  printf("toto%f\n",5.);  


}*/

































