
#include <hip/hip_runtime.h>


//#include "cublas_v2.h"
//#include "cublas.h"
 
//#include <thrust/sort.h>


//#include     "math.h"



#define ACC 40.0
#define BIGNO 1.0e10
#define BIGNI 1.0e-10




__device__ double bessi0( double x )
/*------------------------------------------------------------*/
/* PURPOSE: Evaluate modified Bessel function In(x) and n=0.  */
/*------------------------------------------------------------*/
{
   double ax,ans;
   double y;


   if ((ax=fabs(x)) < 3.75) {
      y=x/3.75,y=y*y;
      ans=1.0+y*(3.5156229+y*(3.0899424+y*(1.2067492
         +y*(0.2659732+y*(0.360768e-1+y*0.45813e-2)))));
   } else {
      y=3.75/ax;
      ans=(exp(ax)/sqrt(ax))*(0.39894228+y*(0.1328592e-1
         +y*(0.225319e-2+y*(-0.157565e-2+y*(0.916281e-2
         +y*(-0.2057706e-1+y*(0.2635537e-1+y*(-0.1647633e-1
         +y*0.392377e-2))))))));
   }
   return ans;
}




__device__ double bessi1( double x)
/*------------------------------------------------------------*/
/* PURPOSE: Evaluate modified Bessel function In(x) and n=1.  */
/*------------------------------------------------------------*/
{
   double ax,ans;
   double y;


   if ((ax=fabs(x)) < 3.75) {
      y=x/3.75,y=y*y;
      ans=ax*(0.5+y*(0.87890594+y*(0.51498869+y*(0.15084934
         +y*(0.2658733e-1+y*(0.301532e-2+y*0.32411e-3))))));
   } else {
      y=3.75/ax;
      ans=0.2282967e-1+y*(-0.2895312e-1+y*(0.1787654e-1
         -y*0.420059e-2));
      ans=0.39894228+y*(-0.3988024e-1+y*(-0.362018e-2
         +y*(0.163801e-2+y*(-0.1031555e-1+y*ans))));
      ans *= (exp(ax)/sqrt(ax));
   }
   return x < 0.0 ? -ans : ans;
}




__device__ double bessi( int n, double x)
/*------------------------------------------------------------*/
/* PURPOSE: Evaluate modified Bessel function In(x) for n >= 0*/
/*------------------------------------------------------------*/
{

   int j;
   double bi,bim,bip,tox,ans;
    
   //I added this line because when n is integer --> In(x)=I|n|(x)
   n=abs(n);
   
   if (n<0)
    return(0);
   
   if (n == 0)
      return( bessi0(x) );
   if (n == 1)
      return( bessi1(x) );


   if (x == 0.0)
      return 0.0;
   else {
      tox=2.0/fabs(x);
      bip=ans=0.0;
      bi=1.0;
      for (j=2*(n+(int) sqrt(ACC*n));j>0;j--) {
         bim=bip+j*tox*bi;
         bip=bi;
         bi=bim;
         if (fabs(bi) > BIGNO) {
            ans *= BIGNI;
            bi *= BIGNI;
            bip *= BIGNI;
         }
         if (j == n) ans=bip;
      }
      ans *= bessi0(x)/bi;
      return  x < 0.0 && n%2 == 1 ? -ans : ans;
   }
}









extern "C"
__global__ void vec_initIndex(int n, int *index)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        index[id] = id;
    }
}






extern "C"
__global__ void vec_computeLocalMaxima(int n, int *localMaxima,int sizeImage,float *input,int sizeFilt,int edgeSizeReject)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
		localMaxima[id]=0;
		int sizeSquare=sizeImage*sizeImage;
		int z=id/sizeSquare;
		int p=id%sizeSquare;
		int x=p/sizeImage;
		int y=p%sizeImage;
		int sizeReject=max(sizeFilt,edgeSizeReject);
		if ((x-sizeReject>=0)&&(y-sizeReject>=0)&&(x+sizeReject<sizeImage)&&(y+sizeReject<sizeImage)){//not edges
			int a,aa,neighbor;
			int ok=1;
			for (a=-sizeFilt;a<=sizeFilt;a++){
				for (a=-sizeFilt;a<=sizeFilt;a++){
					neighbor=z*sizeSquare+(x+a)*sizeImage+(y+aa);
					if (input[id]<input[neighbor]){
						ok=0;
						goto next;
					}
						
				}
			}
			next:
			if (ok==1){
				localMaxima[id]=1;
			}
		}
		
    }
}





extern "C"
__global__ void vec_eraseNonLocalMaxima(int n, float *input,int *localMaxima)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
		if (localMaxima[id]==0){
			input[id]=-1;
		}
		
    }
}







extern "C"
__global__ void vec_set (int n, double *result, double  value)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = value;
    }
}


//=== Vector arithmetic ======================================================

extern "C"
__global__ void vec_add (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] + y[id];
    }
}


extern "C"
__global__ void vec_sub (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] - y[id];
    }
}


extern "C"
__global__ void vec_subFloat (int n, float *result, float  *x, float  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] - y[id];
    }
}


extern "C"
__global__ void vec_mul (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] * y[id];
    }
}



extern "C"
__global__ void vec_mul_fl (int n, float *result, float  *x, float  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] * y[id];
    }
}





extern "C"
__global__ void vec_mul_fl_pow (int n, float *result, float  *x, float  *y_p,float power)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        if (y_p[id]>0){
            int count=0;
            do{
                result[id] = (float)(pow((double)y_p[id],(double)power) * (double)x[id]);
                power/=2;
                count++;
            }while((isnan(result[id]))&&(count<20));
            
            if (isnan(result[id])){
                result[id] = x[id];
                //printf("NAN value %f\n",result[id]);
            }
        }
        else{
            result[id] = x[id];
        }
        
    }
}



extern "C"
__global__ void vec_div (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] / y[id];
    }
}

extern "C"
__global__ void vec_negate (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = -x[id];
    }
}




//=== Vector-and-scalar arithmetic ===========================================

extern "C"
__global__ void vec_addScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] + y;
    }
}

extern "C"
__global__ void vec_addScalarFloat (int n, float *result, float  *x, float  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] + y;
    }
}


extern "C"
__global__ void vec_subScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] - y;
    }
}


extern "C"
__global__ void vec_mulScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] * y;
    }
}


extern "C"
__global__ void vec_mulScalarFloat (int n, float *result, float  *x, float  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] * y;
    }
}


extern "C"
__global__ void vec_divScalar (int n, double *result, double  *x, double  y)
{

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x[id] / y;
    }
}




extern "C"
__global__ void vec_scalarAdd (int n, double *result, double  x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x + y[id];
    }
}


extern "C"
__global__ void vec_scalarSub (int n, double *result, double  x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x - y[id];
    }
}


extern "C"
__global__ void vec_scalarMul (int n, double *result, double  x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x * y[id];
    }
}


extern "C"
__global__ void vec_scalarDiv (int n, double *result, double  x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = x / y[id];
    }
}









//=== Vector comparison ======================================================

extern "C"
__global__ void vec_lt (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] < y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_lte (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] <= y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_eq (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] == y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gte (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] >= y[id])?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gt (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] > y[id])?1.0:0.0;
    }
}



extern "C"
__global__ void vec_ne (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] != y[id])?1.0:0.0;
    }
}




//=== Vector-and-scalar comparison ===========================================

extern "C"
__global__ void vec_ltScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] < y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_lteScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] <= y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_eqScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] == y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gteScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] >= y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_gtScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] > y)?1.0:0.0;
    }
}


extern "C"
__global__ void vec_neScalar (int n, double *result, double  *x, double  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (x[id] != y)?1.0:0.0;
    }
}











//=== Vector math (one argument) =============================================


// Calculate the arc cosine of the input argument.
extern "C"
__global__ void vec_acos (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = acos(x[id]);
    }
}


// Calculate the nonnegative arc hyperbolic cosine of the input argument.
extern "C"
__global__ void vec_acosh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = acosh(x[id]);
    }
}


// Calculate the arc sine of the input argument.
extern "C"
__global__ void vec_asin (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = asin(x[id]);
    }
}


// Calculate the arc hyperbolic sine of the input argument.
extern "C"
__global__ void vec_asinh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = asinh(x[id]);
    }
}


// Calculate the arc tangent of the input argument.
extern "C"
__global__ void vec_atan (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = atan(x[id]);
    }
}


// Calculate the arc hyperbolic tangent of the input argument.
extern "C"
__global__ void vec_atanh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = atanh(x[id]);
    }
}


// Calculate the cube root of the input argument.
extern "C"
__global__ void vec_cbrt (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = cbrt(x[id]);
    }
}


// Calculate ceiling of the input argument.
extern "C"
__global__ void vec_ceil (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = ceil(x[id]);
    }
}


// Calculate the cosine of the input argument.
extern "C"
__global__ void vec_cos (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = cos(x[id]);
    }
}


// Calculate the hyperbolic cosine of the input argument.
extern "C"
__global__ void vec_cosh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = cosh(x[id]);
    }
}


// Calculate the cosine of the input argument × p .
extern "C"
__global__ void vec_cospi (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = cospi(x[id]);
    }
}


// Calculate the complementary error function of the input argument.
extern "C"
__global__ void vec_erfc (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = erfc(x[id]);
    }
}


// Calculate the inverse complementary error function of the input argument.
extern "C"
__global__ void vec_erfcinv (int n, double *result, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = erfcinv(y[id]);
    }
}


// Calculate the scaled complementary error function of the input argument.
extern "C"
__global__ void vec_erfcx (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = erfcx(x[id]);
    }
}


// Calculate the error function of the input argument.
extern "C"
__global__ void vec_erf (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = erf(x[id]);
    }
}


// Calculate the inverse error function of the input argument.
extern "C"
__global__ void vec_erfinv (int n, double *result, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = erfinv(y[id]);
    }
}


// Calculate the base 10 exponential of the input argument.
extern "C"
__global__ void vec_exp10 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = exp10(x[id]);
    }
}


// Calculate the base 2 exponential of the input argument.
extern "C"
__global__ void vec_exp2 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = exp2(x[id]);
    }
}


// Calculate the base e exponential of the input argument.
extern "C"
__global__ void vec_exp (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = exp(x[id]);
    }
}


// Calculate the base e exponential of the input argument, minus 1.
extern "C"
__global__ void vec_expm1 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = expm1(x[id]);
    }
}


// Calculate the absolute value of its argument.
extern "C"
__global__ void vec_fabs (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fabs(x[id]);
    }
}


// Calculate the largest integer less than or equal to x.
extern "C"
__global__ void vec_floor (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = floor(x[id]);
    }
}


// Calculate the value of the Bessel function of the first kind of order 0 for the input argument.
extern "C"
__global__ void vec_j0 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = j0(x[id]);
    }
}


// Calculate the value of the Bessel function of the first kind of order 1 for the input argument.
extern "C"
__global__ void vec_j1 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = j1(x[id]);
    }
}


// Calculate the natural logarithm of the absolute value of the gamma function of the input argument.
extern "C"
__global__ void vec_lgamma (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = lgamma(x[id]);
    }
}


// Calculate the base 10 logarithm of the input argument.
extern "C"
__global__ void vec_log10 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = log10(x[id]);
    }
}


// Calculate the value of l o g e ( 1 + x ) .
extern "C"
__global__ void vec_log1p (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = log1p(x[id]);
    }
}


// Calculate the base 2 logarithm of the input argument.
extern "C"
__global__ void vec_log2 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = log2(x[id]);
    }
}


// Calculate the doubleing point representation of the exponent of the input argument.
extern "C"
__global__ void vec_logb (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = logb(x[id]);
    }
}


// Calculate the natural logarithm of the input argument.
extern "C"
__global__ void vec_log (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = log(x[id]);
    }
}


// Calculate the standard normal cumulative distribution function.
extern "C"
__global__ void vec_normcdf (int n, double *result, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = normcdf(y[id]);
    }
}


// Calculate the inverse of the standard normal cumulative distribution function.
extern "C"
__global__ void vec_normcdfinv (int n, double *result, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = normcdfinv(y[id]);
    }
}


// Calculate reciprocal cube root function.
extern "C"
__global__ void vec_rcbrt (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = rcbrt(x[id]);
    }
}


// Round input to nearest integer value in doubleing-point.
extern "C"
__global__ void vec_rint (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = rint(x[id]);
    }
}


// Round to nearest integer value in doubleing-point.
extern "C"
__global__ void vec_round (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = round(x[id]);
    }
}


// Calculate the reciprocal of the square root of the input argument.
extern "C"
__global__ void vec_rsqrt (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = rsqrt(x[id]);
    }
}


// Calculate the sine of the input argument.
extern "C"
__global__ void vec_sin (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = sin(x[id]);
    }
}


// Calculate the hyperbolic sine of the input argument.
extern "C"
__global__ void vec_sinh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = sinh(x[id]);
    }
}


// Calculate the sine of the input argument × p .
extern "C"
__global__ void vec_sinpi (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = sinpi(x[id]);
    }
}


// Calculate the square root of the input argument.
extern "C"
__global__ void vec_sqrt (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = sqrt(x[id]);
    }
}


// Calculate the tangent of the input argument.
extern "C"
__global__ void vec_tan (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = tan(x[id]);
    }
}


// Calculate the hyperbolic tangent of the input argument.
extern "C"
__global__ void vec_tanh (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = tanh(x[id]);
    }
}


// Calculate the gamma function of the input argument.
extern "C"
__global__ void vec_tgamma (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = tgamma(x[id]);
    }
}


// Truncate input argument to the integral part.
extern "C"
__global__ void vec_trunc (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = trunc(x[id]);
    }
}


// Calculate the value of the Bessel function of the second kind of order 0 for the input argument.
extern "C"
__global__ void vec_y0 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = y0(x[id]);
    }
}


// Calculate the value of the Bessel function of the second kind of order 1 for the input argument.
extern "C"
__global__ void vec_y1 (int n, double *result, double  *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = y1(x[id]);
    }
}











//=== Vector math (two arguments) ============================================





// Create value with given magnitude, copying sign of second value.
extern "C"
__global__ void vec_copysign (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = copysign(x[id], y[id]);
    }
}

// Compute the positive difference between x and y.
extern "C"
__global__ void vec_fdim (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fdim(x[id], y[id]);
    }
}

// Divide two doubleing point values.
extern "C"
__global__ void vec_fdivide (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fdivide(x[id], y[id]);
    }
}

// Determine the maximum numeric value of the arguments.
extern "C"
__global__ void vec_fmax (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fmax(x[id], y[id]);
    }
}

// Determine the minimum numeric value of the arguments.
extern "C"
__global__ void vec_fmin (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fmin(x[id], y[id]);
    }
}

// Calculate the doubleing-point remainder of x / y.
extern "C"
__global__ void vec_fmod (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = fmod(x[id], y[id]);
    }
}

// Calculate the square root of the sum of squares of two arguments.
extern "C"
__global__ void vec_hypot (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = hypot(x[id], y[id]);
    }
}

// Return next representable single-precision doubleing-point value afer argument.
extern "C"
__global__ void vec_nextafter (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = nextafter(x[id], y[id]);
    }
}

// Calculate the value of first argument to the power of second argument.
extern "C"
__global__ void vec_pow (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = pow(x[id], y[id]);
    }
}

// Compute single-precision doubleing-point remainder.
extern "C"
__global__ void vec_remainder (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = remainder(x[id], y[id]);
    }
}















extern "C"
__global__ void vec_testkernel (int n, double *result, double  *x, double  *y)
{
    
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	for (int j=0;j<100;j++){//stupid...just to test speed
	    result[id] = x[id] * y[id]+sqrt(pow(3.14159,id));;;
	}



}









extern "C"
__global__ void vec_computePSF_phase (int n, double *realOutput, double *imagOutput, double  *kx, double  *ky, double  *kz, double  *pupil, double  *phase,double dx, double dy, double dz)
{
    double x;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        x= kx[id]*dx + ky[id]*dy + kz[id]*dz + phase[id];
		realOutput[id]=pupil[id]*cos(x);
		imagOutput[id]=pupil[id]*sin(x);
		//realOutput[id]=phase[id];
		//imagOutput[id]=sin(phase[id]);
    }



}



extern "C"
__global__ void vec_computePSF_phaseN (int n, double  *kx, double  *ky, double  *kz, double  *pupil, double  *phase,double dx, double dy, double dz, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, double *fft)
{
    double x;

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        x= kx[id]*dx + ky[id]*dy + kz[id]*dz + phase[id];
		
		fft[sparseIndexEvenDisk[id]]=pupil[id]*cos(x);
		fft[sparseIndexOddDisk[id]]=pupil[id]*sin(x);


		
    }



}





extern "C"
__global__ void vec_computePSF_phaseNwithOil (int n, double  *kx, double  *ky, double  *kz,double  *kz_is_imag, double  *kz_oil,double  *kz_oil_is_imag, double  *pupil, double  *phase,double dx, double dy, double dz, double dz_oil, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, double *fft)
{
    double x,y,z;

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		z= kx[id]*dx + ky[id]*dy + phase[id];
        x= z + kz[id]*dz - kz_oil[id]*dz_oil;
		y= z + kz[id]*dz*kz_is_imag[id] - kz_oil[id]*dz_oil*kz_oil_is_imag[id];
		fft[sparseIndexEvenDisk[id]]=pupil[id]*cos(x);
		fft[sparseIndexOddDisk[id]]=pupil[id]*sin(y);


		
    }



}




extern "C"         
__global__ void vec_computePSF_phaseNMany (int n,int sizePart,int sizeTot, double  *kx, double  *ky, double  *kz, double  *pupil, double  *phase,double* position, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, double *fft,int many)
{
    double x;
	
	int u,p;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
			u=id/sizePart;
			p=id%sizePart;
		    x= kx[p]*position[u] + ky[p]*position[u+many] + kz[p]*position[u+2*many] + phase[p];
			fft[sparseIndexEvenDisk[p]+u*sizeTot]=pupil[p]*cos(x);
			fft[sparseIndexOddDisk[p]+u*sizeTot]=pupil[p]*sin(x);
			
			

    }



}






//put psf into image
//n=widthImage*heightImage
extern "C"         
__global__ void vec_computeModelAndLikelihood(int n,int widthImage,int heightImage,int widthPSF,int heightPSF, int numberPSF,double  *likelihood,double  *model,float  *image, float  *psf, float  *parameters,int index_x_parameter, int index_y_parameter,int index_photon_parameter,float  *background)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	
	int x=id/heightImage;
	int y=id%heightImage;
	if (id < n)
    {
        int x_start,y_start;
        int X,Y;
        model[id]=(double)background[id];//we should put bckg here
        for (int index_psf=0;index_psf<numberPSF;index_psf++){
			x_start=parameters[index_x_parameter*numberPSF+index_psf];
			X=x-x_start;
			if ((X>=0)&&(X<widthPSF)){
			    y_start=parameters[index_y_parameter*numberPSF+index_psf];
			    Y=y-y_start;
			    if ((Y>=0)&&(Y<heightPSF)){
			        model[id]+=(double)psf[index_psf*widthPSF*heightPSF+X*heightPSF+Y]*(double)parameters[index_photon_parameter*numberPSF+index_psf];
		        }
		        
		    }
		}
		
		if (model[id]>0)
			likelihood[id]=model[id]-(double)image[id]*log(model[id]);
		else
			likelihood[id]=0;

    }



}





//put psf into image
//n=widthImage*heightImage
extern "C"         
__global__ void vec_computeLikelihoodAndModelwithPhotonNumberAndBackground(int n, int numberPSF,double  *likelihood,float  *image,double  *model, float  *psf, float  *parameters,int index_photon_parameter,double *bckg)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int index_psf=id/(n/numberPSF);
	if (id < n)
    {
        //printf("%d    %d    %d \n",index_psf,(index_photon_parameter*numberPSF+index_psf));
        model[id]=(double)psf[id]*(double)parameters[index_photon_parameter*numberPSF+index_psf]+bckg[id];
        if (model[id]>0){
		    likelihood[id]=model[id]-(double)image[id]*log(model[id]);
	    }
	    else{
	        likelihood[id]=0;
        }
    }
}



//put psf into image
//n=widthImage*heightImage
extern "C"         
__global__ void vec_computeModelwithPhotonNumber(int n, int numberPSF,double  *model, float  *psf, float  *parameters,int index_photon_parameter)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int index_psf=id/(n/numberPSF);
	if (id < n)
    {
        //printf("%d    %d    %d \n",index_psf,(index_photon_parameter*numberPSF+index_psf));
        model[id]=(double)psf[id]*(double)parameters[index_photon_parameter*numberPSF+index_psf];
		
    }
}





//put psf into image
//n=widthImage*heightImage
extern "C"         
__global__ void vec_subtractModelwithPhotonNumber(int n, int numberPSF,double  *modelsub,double  *model, float  *psf, float  *parameters,int index_photon_parameter)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int index_psf=id/(n/numberPSF);
	//int index_psf=id%numberPSF;
	if (id < n)
    {
        
        modelsub[id]=model[id]-((double)psf[id]*(double)parameters[index_photon_parameter*numberPSF+index_psf]);
		     
    }
}






//put psf into image
//n=widthImage*heightImage
extern "C"         
__global__ void vec_partialModel(int n,double  *result,double  *a, double  *b, float  h)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        
        result[id]=(a[id]-b[id])/(double)(2*h);
		     
    }
}






//put psf into image
//n=widthImage*heightImage
extern "C"         
__global__ void vec_chi2(long n,int numPixelsPSF,double  *result,double  *model, float  *image)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        if (model[id]>0){
            result[id]=(model[id]-(double)image[id])*(model[id]-(double)image[id])/(model[id]*(double)numPixelsPSF);
        }
        else{
            result[id]=1./(double)numPixelsPSF;
        }
		     
    }
}





//put psf into image
//n=widthImage*heightImage
extern "C"         
__global__ void vec_computeFisherMatrix(int n,int psfsize_square, int numberPSF,float  *fisher,double  *model, double  *modelx,double  *modely,double  *modelz,double  *modelphoton)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	
	if (id < n)
    {
        int idPSF=id/(psfsize_square*16);
	    int xyf=id%(psfsize_square*16);
	    int idFisher=xyf/psfsize_square;
	    int idPixel=xyf%psfsize_square;
	    int u=idFisher/4;
	    int v=idFisher%4;
	    int position=idPSF*psfsize_square+idPixel;
	    //printf("id  %d    idPSF %d   xyf %d     uv %d %d    idPixel %d   position %d\n",id,idPSF,xyf,u,v,idPixel,position);
	    if (model[position]>0){
	    
            fisher[id]=(float)(1/model[position]);
            if (u==0){
                fisher[id]*=(float)modelx[position];
            }
            else if (u==1){
                fisher[id]*=(float)modely[position];
            }
            else if (u==2){
                fisher[id]*=(float)modelz[position];
            }
            else if (u==3){
                fisher[id]*=(float)modelphoton[position];
            }
            
            if (v==0){
                fisher[id]*=(float)modelx[position];
            }
            else if (v==1){
                fisher[id]*=(float)modely[position];
            }
            else if (v==2){
                fisher[id]*=(float)modelz[position];
            }
            else if (v==3){
                fisher[id]*=(float)modelphoton[position];
            }
            
            
            
        }
        else{
            fisher[id]=0;
        }
		     
    }
}





//does the reverse of vec_computeModelAndLikelihood: put image in PSF stack (not useful)
extern "C"         
__global__ void vec_cropFromImage(int n,int widthPSF,int heightPSF, int numberPSF,double  *result,int widthImage,int heightImage, double  *image, float  *parameters,int index_x_parameter, int index_y_parameter)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int index_psf=id%numberPSF;
	int xy=id/numberPSF;
	int x=xy/heightPSF;
	int y=xy%heightPSF;
	if (id < n)
    {
			int x_start=(int)parameters[index_x_parameter*numberPSF+index_psf];
			int y_start=(int)parameters[index_y_parameter*numberPSF+index_psf];
			int X=(x_start+x);
			int Y=y_start+y;
			//printf("%d  %d  / X%d   Y%d     x %d     y %d      xs%d     ys %d   xy %d     index_psf %d\n",id,(xy+index_psf*widthPSF*heightPSF),X,Y,x,y,x_start,y_start,xy,index_psf);
			if ((X>=0)&&(Y>=0)&&(X<widthImage)&&(Y<heightImage)){
			    result[xy+index_psf*widthPSF*heightPSF]=image[X*heightImage+Y];
		    }
		    else{
		        result[xy+index_psf*widthPSF*heightPSF]=0;
		    }
			

    }



}





//does the reverse of vec_computeModelAndLikelihood: put image in PSF stack (not useful)
extern "C"         
__global__ void vec_cropFromImageFloat(int n,int widthPSF,int heightPSF, int numberPSF,float  *result,int widthImage,int heightImage, float  *image, float  *parameters,int index_x_parameter, int index_y_parameter)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int index_psf=id%numberPSF;
	int xy=id/numberPSF;
	int x=xy/heightPSF;
	int y=xy%heightPSF;
	if (id < n)
    {
			int x_start=parameters[index_x_parameter*numberPSF+index_psf];
			int y_start=parameters[index_y_parameter*numberPSF+index_psf];
			int X=(x_start+x);
			int Y=y_start+y;
			//printf("%d  %d  / X%d   Y%d     x %d     y %d      xs%d     ys %d   xy %d     index_psf %d\n",id,(xy+index_psf*widthPSF*heightPSF),X,Y,x,y,x_start,y_start,xy,index_psf);
			if ((X>=0)&&(Y>=0)&&(X<widthImage)&&(Y<heightImage)){
			    result[xy+index_psf*widthPSF*heightPSF]=image[X*heightImage+Y];
		    }
		    else{
		        result[xy+index_psf*widthPSF*heightPSF]=0;
		    }
			

    }



}





//does the reverse of vec_computeModelAndLikelihood: put image in PSF stack
extern "C"         
__global__ void vec_shiftParameter(int n,int indexParameter,float h,float *parameters)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
			parameters[indexParameter*n+id]+=h;
    }



}






extern "C"         
__global__ void vec_updateParameter(int n,int indexParameter,float h,double *lik1,double *lik2,double *lik3,float *parameters,float *parameterSave,float *gamma_weight,float minJump,float maxJump)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        if (gamma_weight[indexParameter*n+id]>0){
			double hh=(double)h;
            double grad;
            if (abs((lik3[id]+lik1[id]-2.*lik2[id])/(hh*hh))==0){
                grad=((lik3[id]-lik1[id])*(2.*hh));
            }
            else{
                grad=((lik3[id]-lik1[id])/(2.*hh))/abs((lik3[id]+lik1[id]-2.*lik2[id])/(hh*hh));
            }
            if (grad>0){
                grad=min(abs((double)maxJump),grad);
                grad=max(abs((double)minJump),grad);
            }
            else{
                grad=max(-abs((double)maxJump),grad);
                grad=min(-abs((double)minJump),grad);
            }
            //if ((indexParameter==2)&&(id==0))
            //    printf("grad   %d     %d    %f     lik  %f    %f    %f   %f\n",indexParameter,id,grad,lik1[id],lik2[id],lik3[id],gamma_weight[indexParameter*n+id]);
            parameterSave[id]=parameters[indexParameter*n+id];
            parameters[indexParameter*n+id]-=(float)grad*gamma_weight[indexParameter*n+id];
        }
            
    }



}




extern "C"         
__global__ void vec_checkLikelihood(int n,int indexParameter,double *likold,double *liknew,float *parameters,float * parameterSave,float *gamma_weight)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
			if (liknew[id]>likold[id]){//if new likelihood not better: change weight and put back weight
                parameters[indexParameter*n+id]=parameterSave[id];
                gamma_weight[indexParameter*n+id]/=10.;
            }
            else{//if new likelihood better: no change anymore
                likold[id]=liknew[id];
                gamma_weight[indexParameter*n+id]=0;
                parameterSave[id]=parameters[indexParameter*n+id];
            }
    }



}





extern "C"         
__global__ void vec_computePSF_phaseNMany_f (int n,int sizePart,int sizeTot, float  *kx, float  *ky, float  *kz, float  *pupil, float  *phase,float* position, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, float *fft,int many)
{
    float x;
	
	int u,p;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
			u=id/sizePart;
			p=id%sizePart;
		    x= kx[p]*position[u] + ky[p]*position[u+many] + kz[p]*position[u+2*many] + phase[p];
			fft[sparseIndexEvenDisk[p]+u*sizeTot]=pupil[p]*cos(x);
			fft[sparseIndexOddDisk[p]+u*sizeTot]=pupil[p]*sin(x);
			
			

    }



}




extern "C"         
__global__ void vec_computePSF_phaseNManywithOil_f (int n,int sizePart,int sizeTot, float  *kx, float  *ky, float  *kz,float  *kz_is_imag, float  *kz_oil,float  *kz_oil_is_imag, float  *pupil, float  *phase,float* position, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, float *fft,int many)
{
    float x,y,z;
	//float x;
	
	int u,p;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
			u=id/sizePart;
			p=id%sizePart;

			//x= kx[p]*position[u] + ky[p]*position[u+many] + phase[p] + kz[p]*position[u+2*many]*kz_is_imag[p] - kz_oil[p]*position[u+3*many]*kz_oil_is_imag[p];
			//fft[sparseIndexEvenDisk[p]+u*sizeTot]=pupil[p]*cos(x);
			//fft[sparseIndexOddDisk[p]+u*sizeTot]=pupil[p]*sin(x);
			
			
			z= kx[p]*position[u] + ky[p]*position[u+many] + phase[p];
		    x= z + kz[p]*position[u+2*many] - kz_oil[p]*position[u+3*many];
			y= z + kz[p]*position[u+2*many]*kz_is_imag[p] - kz_oil[p]*position[u+3*many]*kz_oil_is_imag[p];
			fft[sparseIndexEvenDisk[p]+u*sizeTot]=pupil[p]*cos(x);
			fft[sparseIndexOddDisk[p]+u*sizeTot]=pupil[p]*sin(y);
			
			

    }



}



extern "C"
__global__ void vec_addFloat(int n, float *result, float  *a, float *b)
{

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = a[id] + b[id];
    }
}



extern "C"
__global__ void vec_addanddivide(int n, float *result, float  *num, float *div, float *added2div)
{//perform num/(div+added2div)

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        div[id]+=added2div[id];
        result[id] = num[id] / (div[id]);
    }
}








extern "C"
__global__ void vec_computeLikelihoodDeconvolution(int n, float *result, float  *I, float *M)
{//perform num/(div+added2div)

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = M[id]-I[id]*log(M[id]);
    }
}







__device__ double bessiRatio( int n, double x)
/*------------------------------------------------------------*/
/* PURPOSE: Evaluate modified Bessel function In-1(x)/In(x) for n >= 0*/
/*------------------------------------------------------------*/
{
    n=abs(n);
    double Y=(x/2)*(x/2);
    double B_up=1;
    double BB_up=1;
    double B_down=1;
    double BB_down=1;
    double k=0;
    double n_up=n-1;
    double n_down=n;
    double ratio=1;
    double iter=sqrt(ACC*n_down);
    int nbIter=0;
    iter=5;
    for (k=1;k<iter;k++){
        B_up*=Y/(k*(k+n_up));
        BB_up+=B_up;
        B_down*=Y/(k*(k+n_down));
        BB_down+=B_down;
        if (B_up>1e+300){
            //break;
        }
        ratio=BB_up/BB_down;
        nbIter++;
    }
    //printf("BB %f  %f   /   %f  %f  / Y %f   ratio : %f  it:%d\n",n_up,n_down,B_up,B_down,Y,ratio,nbIter);
    printf("BB %f  %f   /   %f         bu:%f bd:%f r:%f \n",n_down,x,(n_down*2/x)*ratio,BB_up,BB_down,ratio);
    return (n_down*2/x)*ratio;

}





//See: On the Computation of Modified Bessel Function Ratios, Mathematics of Computation, September 1978
__device__ double R( double * n, double * x,double k,double * K){
    
    if (k>=*K){
        return 0.;
    }
    else{
        return 1./( (2./ *x)*(*n+k) + R(n,x,k+1,K));
    }
    
}

__device__ double myBessiRatio( double n, double x)
/*------------------------------------------------------------*/
/* PURPOSE: Evaluate modified Bessel function In-1(x)/In(x) for n >= 0*/
/*------------------------------------------------------------*/
{
    n=abs(n);
    
    double iterNumber=20;
    
    return 1./R(&n,&x,0,&iterNumber);

}


__device__ double myBessiRatioNonRecusrsive( double n, double x)
/*------------------------------------------------------------*/
/* PURPOSE: Evaluate modified Bessel function In-1(x)/In(x) for n >= 0*/
/*------------------------------------------------------------*/
{
    
    
    double som=0;
    for (double k=50;k>=0;k--){
        som+=((2./x)*(n+k));
        som=1./som;
    }
    return 1./som;
}


//Even if it is float, we assume I is photon count here --> we cast it to integer
//Perform Ib-Ia skellam
// n should be full size
// minReplacement = 1 by default: When I<<M --> b1==inf  --> inf value So we replace "inf" values by "minReplacement" in result
// maxReplacement = 1 by default: When I>>M --> b1==0  --> division Nan So we replace "Nan" values by "maxReplacement" in result
extern "C"
__global__ void vec_skellam_order1(long n, long imageSize,float *result, float  *I, float *M)
{//perform num/(div+added2div)

    long idx = threadIdx.x + blockIdx.x * blockDim.x;
	long idy = threadIdx.y + blockIdx.y * blockDim.y;
	long id = idy * gridDim.x * blockDim.x + idx;
	int indexFrame=id/imageSize;
	long indexImage=id%imageSize;
	float sqrtMaMb,tmp,Itmp,tt;
	//float Mb,Ma;
	//float Ib,Ia;
	//double b0,b1;
	
    if (id < n)
    {
    
        if (id<imageSize){
            result[id+n*2]=0;
            /*Itmp=I[indexImage+imageSize]-I[indexImage];
            result[id+n*2]=Itmp/(2*M[indexImage+imageSize]);*/
        }
        if (id>=(n-imageSize)){
            result[id+n]=0;
            /*Itmp=I[indexImage+imageSize]-I[indexImage];
            result[id+n]=-Itmp/(2*M[indexImage]);*/
        }
        //Ma=M[indexImage];
        //Mb=M[indexImage+imageSize];
        //Ia=I[indexImage];
        //Ib=I[indexImage+imageSize];
        int index0=indexImage+imageSize*(indexFrame);//current frame
        int index1=indexImage+imageSize*(indexFrame+1);//next frame
        if (id<(n-imageSize)){// because we do it nbFrame-1 times
            Itmp=I[index1]-I[index0];
            sqrtMaMb=(float)sqrt(M[index0]*M[index1]);
            tt=sqrtMaMb*2;
            //printf("test %f   %f   \n",Itmp,tt);
            
            tmp= myBessiRatioNonRecusrsive((double)abs(Itmp),(double)tt)    ;
            
            /*//here, we compute skellam :
            b1=bessi( (int)Itmp, (double)tmp);
            b0=bessi( (int)(Itmp-1), (double)tmp);
            //printf("b0 %f   b1 %f   tmp %f Itmp %f     sqrtMaMb %f     %f   bool: %d\n",b0,b1,tmp,Itmp,sqrtMaMb,1e+30,(b1>1e+30));
            if (b1<1e-307){//10E-30
                //result[offset+id+n*2]=maxReplacement;
                //result[offset+id+n*3]=maxReplacement;
                result[id+n]=maxReplacement;
                result[id+n*2+imageSize]=maxReplacement;
            }
            else if (b0>1e+307){//10E+36
                //result[offset+id+n*2]=minReplacement;
                //result[offset+id+n*3]=minReplacement;
                result[id+n]=minReplacement;
                result[id+n*2+imageSize]=minReplacement;
            }
            else{
                tmp=(double)((double)1./(double)sqrtMaMb)*( b0  -  ( (double)(Itmp/tmp) * b1 ) )/b1;*/
                
                //result[offset+id+n*2]=(float)(M[indexImage+imageSize]*tmp);
                //result[offset+id+n*3]=(float)(M[indexImage]*tmp);
                result[id+n]=(float)(M[index1]*( tmp- (double)(abs(Itmp)/tt) ))/(double)sqrtMaMb;
                result[id+n*2+imageSize]=(float)(M[index0]*( tmp- (double)(abs(Itmp)/tt) )/(double)sqrtMaMb);
            //}
            
            //result[offset+id]=-Itmp/(2*M[indexImage]);//  (Ia-Ib)/2Ma
            //result[offset+id+n]=Itmp/(2*M[indexImage+imageSize]);//  (Ib-Ia)/2Mb
            result[id+n]+=-Itmp/(2*M[index0]);//  (Ia-Ib)/2Ma
            result[id+n*2+imageSize]+=Itmp/(2*M[index1]);//  (Ib-Ia)/2Mb
            
            if (index0/imageSize!=0){//first image
                result[id+n]/=2;
                
            }
            if (index1/imageSize!=((n/imageSize)-1)){
                result[id+n*2+imageSize]/=2;
            }
            
        }
    }
}






extern "C"         
__global__ void vec_thetest(int n,float  *kz_is_imag)
{
    
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
			
printf("kz -> %d  %f  \n",id,kz_is_imag[id]);
			
			

    }



}



//WARNING : device_sum size should be gridDim.x
__device__ void sumTheBlocks (double *device_sum)
{

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < 1)//lose few time... i agree
    {
		for (int i=1;i<gridDim.x;i++){
			device_sum[0]+=device_sum[i];
		}
		
	}
}


__device__ int truc;
__device__ int barrier;
__device__ int barrier2;
__device__ void initSynchroBlocks(){
	if (threadIdx.x + blockIdx.x * blockDim.x==0){
		barrier=gridDim.x;
		barrier2=gridDim.x;
	}
}

__device__ void synchroBlocks(){
	__syncthreads();
	if (threadIdx.x==0){
		atomicSub( &barrier , 1 );
	}
	//atomicAdd( &truc , 1 );
	//if ( threadIdx.x == 0 )
        //while ( atomicCAS( &barrier , 0 , 0 ) != 0 );//does not work: infinite loop...

    __syncthreads();
}
__device__ void synchroBlocks2(){
	__syncthreads();
	if (threadIdx.x==0){
		atomicSub( &barrier2 , 1 );
	}
	//atomicAdd( &truc , 1 );
	//if ( threadIdx.x == 0 )
        //while ( atomicCAS( &barrier2 , 0 , 0 ) != 0 );//does not work: infinite loop...

    __syncthreads();
}








extern "C"
__device__ void vec_dense2Sparse (int n, double *device_input, int *device_sparse, double *device_output)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		device_output[id]=device_input[device_sparse[id]];
    }
}


extern "C"
__device__ void vec_sparse2Dense (int n, double *device_input, int *device_sparse, double *device_output)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		device_output[device_sparse[id]]=device_input[id];
    }
}



extern "C"
__global__ void vec_computePSF_signal (int n, double *result, double *real, double  *imag,double divide)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        x=real[id]/divide;
		y=imag[id]/divide;
		result[id]=x*x+y*y;
    }



}




extern "C"
__global__ void vec_computePSF_signalN (int n, double *result, double divide, int *sparseIndexEvenShiftOutput, int *sparseIndexOddShiftOutput, double *fft)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		x=fft[sparseIndexEvenShiftOutput[id]]/divide;
		y=fft[sparseIndexOddShiftOutput[id]]/divide;
		result[id]=x*x+y*y;
    }



}


extern "C"
__global__ void vec_computePSF_signalN2 (int n, double divide, int *sparseIndexEvenShiftOutput, int *sparseIndexOddShiftOutput, double *fft,int *sparseIndexEvenShiftOutputNext,int *sparseIndexOddShiftOutputNext, double *psffft)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		x=fft[sparseIndexEvenShiftOutput[id]]/divide;
		y=fft[sparseIndexOddShiftOutput[id]]/divide;
		psffft[sparseIndexEvenShiftOutputNext[id]]=x*x+y*y;
		psffft[sparseIndexOddShiftOutputNext[id]]=0;

		//psffft[id]=x*x+y*y;
    }



}





extern "C"
__global__ void vec_computePSF_signalN2Many (int n,int sizePart,int sizeTot, double divide, int *sparseIndexEvenShiftOutput, int *sparseIndexOddShiftOutput, double *fft,int *sparseIndexEvenShiftOutputNext,int *sparseIndexOddShiftOutputNext, double *psffft)
{
	int u,p;
	
	
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		u=id/sizePart;
		p=id%sizePart;
		x=fft[sparseIndexEvenShiftOutput[p]+sizeTot*u]/divide;
		y=fft[sparseIndexOddShiftOutput[p]+sizeTot*u]/divide;
		psffft[sparseIndexEvenShiftOutputNext[p]+2*sizePart*u]=x*x+y*y;
		psffft[sparseIndexOddShiftOutputNext[p]+2*sizePart*u]=0;

		//psffft[id]=x*x+y*y;
    }



}






extern "C"
__global__ void vec_computePSF_signalN2Many_f (int n,int sizePart,int sizeTot, float divide, int *sparseIndexEvenShiftOutput, int *sparseIndexOddShiftOutput, float *fft,int *sparseIndexEvenShiftOutputNext,int *sparseIndexOddShiftOutputNext, float *psffft)
{
	int u,p;
	
	
    float x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		u=id/sizePart;
		p=id%sizePart;
		x=fft[sparseIndexEvenShiftOutput[p]+sizeTot*u]/divide;
		y=fft[sparseIndexOddShiftOutput[p]+sizeTot*u]/divide;
		psffft[sparseIndexEvenShiftOutputNext[p]+2*sizePart*u]=x*x+y*y;
		psffft[sparseIndexOddShiftOutputNext[p]+2*sizePart*u]=0;

		//psffft[id]=x*x+y*y;
    }



}




extern "C"
__global__ void vec_computePSF_signalsqrt (int n, double *result, double *real, double  *imag,double divide)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        x=real[id]/divide;
		y=imag[id]/divide;
		result[id]=sqrt(x*x+y*y);
    }



}





extern "C"
__global__ void vec_computePSF_signalNsqrt (int n, double *result, double *fft,double divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        x=fft[sparseIndexEvenShift2D[id]]/divide;
		y=fft[sparseIndexOddShift2D[id]]/divide;
		result[id]=sqrt(x*x+y*y);
    }



}




extern "C"
__global__ void vec_computePSF_signalNsqrtMany (int n, int sizeSubImage,double *result, double *fft,double divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D)
{
    double x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
 	int id2=id%sizeSubImage;
	int id3=id/sizeSubImage;
	if (id < n)
    {
        x=fft[sparseIndexEvenShift2D[id2]+id3*sizeSubImage*2]/divide;
		y=fft[sparseIndexOddShift2D[id2]+id3*sizeSubImage*2]/divide;
		result[id]=sqrt(x*x+y*y);
    }



}




extern "C"
__global__ void vec_computePSF_signalNsqrtMany_f (int n, int sizeSubImage,float *result, float *fft,float divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D)
{
    float x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
 	int id2=id%sizeSubImage;
	int id3=id/sizeSubImage;
	if (id < n)
    {
        x=fft[sparseIndexEvenShift2D[id2]+id3*sizeSubImage*2]/divide;
		y=fft[sparseIndexOddShift2D[id2]+id3*sizeSubImage*2]/divide;
		result[id]=sqrt(x*x+y*y);
    }



}




extern "C"
__global__ void vec_computePSF_signalNsqrtMany_fcrop (int n, int sizeSubImage, int sizeSubImageFull,float *result, float *fft,float divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D)
{
    float x,y;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
 	int id2=id%sizeSubImage;
	int id3=id/sizeSubImage;
	int id4=id3*sizeSubImageFull;
	if (id < n)
    {
        x=fft[sparseIndexEvenShift2D[id2]+id4*2]/divide;
		y=fft[sparseIndexOddShift2D[id2]+id4*2]/divide;
		result[id]=sqrt(x*x+y*y);
    }



}

__device__ void divideBySum (int n, double *result, double *tmpsum)
{
    
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	synchroBlocks2();
	if (id < n)
    {	
		result[id]/=tmpsum[0];
	}

}







extern "C"
__global__ void vec_mulMany (int n, int sizeKernel, double *result, double  *x, double  *kernel)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id%sizeKernel;
    if (id < n)
    {
        result[id] = x[id] * kernel[id2];
    }
}



extern "C"
__global__ void vec_divScalarMany (int n,  int sizeSubImage,double *result, double  *x, double  *div)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeSubImage;
    if (id < n)
    {
		if (div[id2]!=0)
        	result[id] = x[id] / div[id2];
    }
}




extern "C"
__global__ void vec_mulMany_f (int n, int sizeKernel, float *result, float  *x, float  *kernel)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id%sizeKernel;
    if (id < n)
    {
        result[id] = x[id] * kernel[id2];
    }
}



extern "C"
__global__ void vec_computeModelMany1 (int n, int sizeImage, double *result, double  *x, double  *amplitude,double background)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
    if (id < n)
    {
        result[id] = x[id] * amplitude[id2] + background;
    }
}


extern "C"
__global__ void vec_computeModelMany2 (int n, int sizeImage, double *result, double  *x, double  *amplitude,double *background)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
    if (id < n)
    {
        result[id] = x[id] * amplitude[id2] + background[id2];
    }
}



//here, background is 2D
extern "C"
__global__ void vec_computeModelMany3 (int n, int sizeImage, double *result, double  *x, double  *amplitude,double *background)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
	int id3=id%sizeImage;
    if (id < n)
    {

        result[id] = x[id] * amplitude[id2] + background[id3];
		
    }
}



extern "C"
__global__ void vec_computeModelMany1_scmos (int n, int sizeImage, double *result, double  *x, double  *amplitude,double background,double  *scmos)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
	int id3=id%sizeImage;
    if (id < n)
    {
        result[id] = x[id] * amplitude[id2] + background + scmos[id3];
    }
}


extern "C"
__global__ void vec_computeModelMany2_scmos (int n, int sizeImage, double *result, double  *x, double  *amplitude,double *background,double  *scmos)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
	int id3=id%sizeImage;
    if (id < n)
    {
        result[id] = x[id] * amplitude[id2] + background[id2] + scmos[id3];
    }
}



//here, background is 2D
extern "C"
__global__ void vec_computeModelMany3_scmos (int n, int sizeImage, double *result, double  *x, double  *amplitude,double *background,double  *scmos)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeImage;
	int id3=id%sizeImage;
    if (id < n)
    {

        result[id] = x[id] * amplitude[id2] + background[id3] + scmos[id3];
		
    }
}



extern "C"
__global__ void vec_divScalarMany_f (int n,  int sizeSubImage,float *result,double *resultDouble, float  *x, float  *div)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeSubImage;
    if (id < n)
    {
		if (div[id2]!=0){
        	result[id] = x[id] / div[id2];
			resultDouble[id] =0;
        	resultDouble[id] =(double)(result[id]);
		}
    }
}








extern "C"
__global__ void vec_computePoissonLikelihood (int n, double *result, double *image, double  *model)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		if (model[id]>0)
			result[id]=model[id]-image[id]*log(model[id]);
		else
			result[id]=10000000;
			
    }



}





extern "C"
__global__ void vec_computeGaussianLikelihood (int n, double *result, double *image, double  *model)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		if (model[id]>0)
			result[id]=(model[id]-image[id])*(model[id]-image[id]);
		else
			result[id]=10000000;
			
    }



}










//reshuffle: 
//exemple 4 PSF to merge in 2 model
//->>> PSF=1,2,3 merged with PSF=4,5,6
//->>> PSF=7,8,9 merged with PSF=10,11,12
//we need to reshuffle here to use then Dgemv for matrix operation
//1,2,3,4,5,6,7,8,9,10,11,12 -> 1,2,3,7,8,9,4,5,6,10,11,12
extern "C"
__global__ void vec_addPhotonsAndBackgroundManyReshuffle (int n, int sizeSubImage,int numberPSFperModel,double *output, double *input, double *photonAndBackground)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;

	if (id < n)
	{
		int idPSF=id/sizeSubImage;
		int idModel=idPSF/numberPSFperModel;
		int idoffset=id%sizeSubImage;
		int idposit=idPSF%numberPSFperModel;
		int idreshuffle=idModel*sizeSubImage +idposit*sizeSubImage*(n/(sizeSubImage*numberPSFperModel))       +idoffset;
		output[idreshuffle]=input[id]*photonAndBackground[idPSF*2]+photonAndBackground[idPSF*2+1];

	}

}






extern "C"
__global__ void vec_addPhotonsAndBackgroundManyReshuffle_scmos (int n, int sizeSubImage,int numberPSFperModel,double *output, double *input, double *photonAndBackground, double * scmos)
{


//print("to do as previous function");



	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	
	if (id < n)
    {
	int idPSF=id/sizeSubImage;
	int idModel=idPSF/numberPSFperModel;
	int idoffset=id%sizeSubImage;
	int idposit=idPSF%numberPSFperModel;
	int idreshuffle=idModel*sizeSubImage +idposit*sizeSubImage*(n/(sizeSubImage*numberPSFperModel))       +idoffset;
	output[idreshuffle]=input[id]*photonAndBackground[idPSF*2]+photonAndBackground[idPSF*2+1]+scmos[id];

    }

}








extern "C"
__global__ void vec_addPhotonsAndBackgroundMany (int n, int sizeSubImage,double *output, double *input, double *photonAndBackground)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeSubImage;
	if (id < n)
    {
		output[id]=input[id]*photonAndBackground[id2*2]+photonAndBackground[id2*2+1];

    }

}






extern "C"
__global__ void vec_addPhotonsAndBackgroundMany_scmos (int n, int sizeSubImage,double *output, double *input, double *photonAndBackground, double * scmos)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeSubImage;
	if (id < n)
    {
		output[id]=input[id]*photonAndBackground[id2*2]+photonAndBackground[id2*2+1]+scmos[id];

    }

}







extern "C"
__global__ void vec_addPhotonsAndBackgroundMany_f (int n, int sizeSubImage,float *output, float *input, float *photonAndBackground)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=id/sizeSubImage;
	if (id < n)
    {
		output[id]=input[id]*photonAndBackground[id2*2]+photonAndBackground[id2*2+1];

    }

}



extern "C"         
__global__ void vec_shrink (int n,float  *output, float  *input, float  threshold)
{
    
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
			if (input[id]<threshold){
			    output[id]=0;
			}
			else{
			    output[id]=input[id];
			}
    }



}





extern "C"
__global__ void vec_double2float (int n, float *output, double *input)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		output[id]=(float)input[id];

    }

}


extern "C"
__global__ void vec_float2double (int n, double *output, float *input)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		output[id]=(double)input[id];

    }

}




//multi kernel complexe conjugate
//*2 because real and imag parts
//n is total size (complex) of kernel divided by 2
//sizeInput is total size (complex) of image divided by 2
extern "C"
__global__ void vec_complexeConjugateKernel (int n,  int sizeInput, float *output, float *input, float *inputKernel)
{
	//n size 
	//int id = 2*(threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = 2*(idy * gridDim.x * blockDim.x + idx);
	int id2=id%(sizeInput*2);
	float real;
	float imag;
	float tmp;
	if (id < n*2)
    {
		real=input[id2]/sqrt((float)sizeInput);
		imag=input[id2+1]/sqrt((float)sizeInput);
		//id : real
		//id+1 : imaginary
		tmp=imag*inputKernel[id+1]+real*inputKernel[id];
		output[id+1]=imag*inputKernel[id]-real*inputKernel[id+1];
		output[id]=tmp;
		

    }

}



extern "C"
__global__ void vec_complexeConjugateKernelSubtract (int n,  int sizeInput, float *output, float *input, float *inputKernel)
{
	//n size 
	//int id = 2*(threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = 2*(idy * gridDim.x * blockDim.x + idx);
	int id2=id%(sizeInput*2);
	float real;
	float imag;
	float tmp;
	if (id < n*2)
    {
		real=input[id2]/sqrt((float)sizeInput);
		imag=input[id2+1]/sqrt((float)sizeInput);
		//id : real
		//id+1 : imaginary
		tmp=imag*inputKernel[id+1]+real*inputKernel[id];
		output[id+1]=imag*inputKernel[id]-real*inputKernel[id+1];
		output[id]=tmp;
		

    }

}





//multi kernel complexe conjugate
//*2 because real and imag parts
//n is total size (complex) of kernel divided by 2
//sizeInput is total size (complex) of image divided by 2
extern "C"
__global__ void vec_complexeMulKernel (int n,  int sizeInput, float *output, float *input, float *inputKernel)
{
	//n size 
	//int id = 2*(threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = 2*(idy * gridDim.x * blockDim.x + idx);
	int id2=id%(n*2);//strange...id2 might be = to id
	float real;
	float imag;
	float tmp;
	if (id < n*2)
    {
		real=input[id2]/sqrt((float)sizeInput);
		imag=input[id2+1]/sqrt((float)sizeInput);
		//id : real
		//id+1 : imaginary
		tmp=real*inputKernel[id]-imag*inputKernel[id+1];
		output[id+1]=imag*inputKernel[id]+real*inputKernel[id+1];
		output[id]=tmp;
		

    }

}





//multi kernel complexe conjugate
//*2 because real and imag parts
//n is total size (complex) of kernel divided by 2
//sizeInput is total size (complex) of image divided by 2
extern "C"
__global__ void vec_complexeMulKernelMany (long n,  long depth,  long size, float *output, float *input, float *inputKernel)
{
	//n size 
	//int id = 2*(threadIdx.x + blockIdx.x * blockDim.x);
	long idx = threadIdx.x + blockIdx.x * blockDim.x;
	long idy = threadIdx.y + blockIdx.y * blockDim.y;
	long idd = 2*(idy * gridDim.x * blockDim.x + idx);
	long id=idd%(2*size*depth);
	
	//int id2=id%(n*2);
	float real;
	float imag;
	float tmp;
	if (idd < n*2)
    {
        
		real=input[idd]/size;
		imag=input[idd+1]/size;
		//id : real
		//id+1 : imaginary
		tmp=real*inputKernel[id]-imag*inputKernel[id+1];
		output[idd+1]=imag*inputKernel[id]+real*inputKernel[id+1];
		output[idd]=tmp;
		

    }

}





extern "C"
__global__ void vec_copyMany(long n, long sizeInput,long depth,long nbFrame, float *output, float *input)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	long idx = threadIdx.x + blockIdx.x * blockDim.x;
	long idy = threadIdx.y + blockIdx.y * blockDim.y;
	long idd = idy * gridDim.x * blockDim.x + idx;
	long id=idd%(sizeInput*depth);
	long idFrame=idd/(sizeInput*depth);
	long idPixel=(id)%(sizeInput);
	//long idZ=(id)/(sizeInput);
	
	if (idd < n)
    {
        long posit=idPixel+idFrame*sizeInput;
		output[idd]=input[posit];
        
        
    }

}




//multi kernel correlation result
//n is total size (complex) of kernel divided by 2
//sizeInput is total size (complex) of image divided by 2
extern "C"
__global__ void vec_mycusparsemoduloSsctrMany(long n, long nbFrame,long depth,long sizeShift, long width,long height,float *output, float *input, int *sparse)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	long idx = threadIdx.x + blockIdx.x * blockDim.x;
	long idy = threadIdx.y + blockIdx.y * blockDim.y;
	long idd = idy * gridDim.x * blockDim.x + idx;
	long sizeSparse=width*height;
	long idFrame=idd/(depth*width*height);
	long id=idd%(depth*width*height);
	long id2=(id)%(sizeSparse);
	long id3=(id)/(sizeSparse);
	if (idd < n)
    {
        
		output[(long)sparse[id2]+sizeShift*id3+idFrame*sizeShift*depth]=input[idd];

    }

}





//multi kernel correlation result
//n is total size (complex) of kernel divided by 2
//sizeInput is total size (complex) of image divided by 2
//process many images at the same times
extern "C"
__global__ void vec_mycusparsemoduloSsctr(int n, int sizeShift, int sizeSparse,float *output, float *input, int *sparse)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=(id)%(sizeSparse);
	int id3=(id)/(sizeSparse);
	if (id < n)
    {
    
		output[sparse[id2]+sizeShift*id3]=input[id];

    }

}





//multi kernel correlation result
//n is total size (complex) of kernel divided by 2
//sizeInput is total size (complex) of image divided by 2
extern "C"
__global__ void vec_makeResultCorrelation(int n, int sizeInput, int sizeFullPadded,float *output, float *input, int *sparse)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=(id)%(sizeInput);
	int id3=(id)/(sizeInput);
	if (id < n)
    {
        
		output[id]=input[sparse[id2]+sizeFullPadded*id3]/sqrt((float)sizeFullPadded/2.);

    }

}


//multi kernel correlation result
//n is total size (complex) of kernel divided by 2
//sizeInput is total size (complex) of image divided by 2
extern "C"
__global__ void vec_normalizeCorrelation(long n, long nbFrame, long depth, long sizeInput,float *output, float *input, float *divide)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	long idx = threadIdx.x + blockIdx.x * blockDim.x;
	long idy = threadIdx.y + blockIdx.y * blockDim.y;
	long idd = idy * gridDim.x * blockDim.x + idx;
	long id=idd%(sizeInput*depth);
	//long idFrame=idd/(sizeInput*depth);
	//long id2=(id)%(sizeInput);
	//long id3=(id)/(sizeInput);
	if (idd < n)
    {
        
		output[idd]=input[idd]/divide[id];

    }

}




//multi kernel correlation result
//n is total size (complex) of kernel divided by 2
//sizeInput is total size (complex) of image divided by 2
extern "C"
__global__ void vec_makeResultCorrelationMany(long n, long nbFrame, long depth, long sizeInput, long sizeFullPadded,float *output, float *input, int *sparse)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	long idx = threadIdx.x + blockIdx.x * blockDim.x;
	long idy = threadIdx.y + blockIdx.y * blockDim.y;
	long idd = idy * gridDim.x * blockDim.x + idx;
	long id=idd%(sizeInput*depth);
	long idFrame=idd/(sizeInput*depth);
	long id2=(id)%(sizeInput);
	long id3=(id)/(sizeInput);
	if (idd < n)
    {
        
		output[idd]=input[sparse[id2]+sizeFullPadded*id3+idFrame*sizeFullPadded*depth];

    }

}




//multi kernel correlation result
//n is total size (complex) of kernel divided by 2
//sizeInput is total size (complex) of image divided by 2
extern "C"
__global__ void vec_turnMatrixMany(long n, long nbFrame, long depth, long sizeInput,float *output, float *input)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	long idx = threadIdx.x + blockIdx.x * blockDim.x;
	long idy = threadIdx.y + blockIdx.y * blockDim.y;
	long idd = idy * gridDim.x * blockDim.x + idx;
	long id=idd%(sizeInput*depth);
	long idFrame=idd/(sizeInput*depth);
	long idPixel=(id)%(sizeInput);
	long idZ=(id)/(sizeInput);
	
	if (idd < n)
    {
        long posit=idPixel+idFrame*sizeInput+idZ*sizeInput*nbFrame;
		output[posit]=input[idd];
        
        
    }

}


extern "C"
__global__ void vec_makeResultCorrelationNormalized(int n, int sizeInput, int sizeFullPadded,float *output, float *input, int *sparse,float divide,float* device_divide,float minValue)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	int id2=(id)%(sizeInput);
	int id3=(id)/(sizeInput);
	if (id < n)
    {
        float tmp=(input[sparse[id2]+sizeFullPadded*id3]/(divide*sqrt((float)sizeFullPadded/2.)))-device_divide[id2];
        if (tmp>0){
		    output[id]=minValue;//tmp;
	    }
	    else{
	        output[id]=minValue;
	    }

    }

}



extern "C"
__global__ void vec_initializeDeconvolution(int n,int nbpsf ,float *o, float *op, float *m,float value)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        o[id]=value;
        op[id]=value;
        if (id<n/nbpsf){
            m[id]=value*(float)nbpsf;
        }

    }

}




extern "C"
__global__ void vec_initializeVectorToValue(int n ,float *v, float value)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        v[id]=value;
        

    }

}

extern "C"
__global__ void vec_chiScore (int n, float *result, float  *image, float  *model)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
        result[id] = (image[id] - model[id])*(image[id] - model[id])/model[id];
    }
}



extern "C"
__global__ void vec_max(int n ,float *v, float value)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
        v[id]=max(v[id],value);
        

    }

}




extern "C"
__global__ void vec_subtractMeanWithSumAsInputWithPositiveConstraint(int n, float *output, float *input, float *sum,float minValue)
{
	//n size 
	//int id = (threadIdx.x + blockIdx.x * blockDim.x);
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	
	if (id < n)
    {
        float tmp=input[id]-(sum[0]/(float)n);
        if (tmp>0){
		    output[id]=tmp;
	    }
	    else{
	        output[id]=minValue;
	    }

    }

}



extern "C"
__global__ void vec_divScalarFloat ( int n, float *result, float  *x, float  y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = idy * gridDim.x * blockDim.x + idx;

	//int offset = threadIdx.x + blockIdx.x * blockDim.x;
    if (offset < n)
    {
        result[offset] = x[offset] / y;
    }
}







extern "C"
__global__ void vec_updateMandOP ( int n, float *m,float *op, float  *x, float  div)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = idy * gridDim.x * blockDim.x + idx;
    float tmp;
	//int offset = threadIdx.x + blockIdx.x * blockDim.x;
    if (offset < n)
    {
        tmp  = m[offset] -  op[offset];
        op[offset] = (x[offset] / div);
        m[offset] = tmp + op[offset];
    }
}





//11 images as input
//25 images as output
//n=3sizesubimages
extern "C"
__global__ void vec_computeCRLB (int n,int sizeMatrix,double *output, double *input,double h)
{
	
	int sizeSubImage=n/(sizeMatrix*sizeMatrix);
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	if (id < n)
    {
		int p=(id/sizeSubImage);//p=0..24
		int positpix=id%sizeSubImage;//0..sizeSubImage-1
		int line=p/(sizeMatrix);//0..4
		int column=p%(sizeMatrix);//0..4
		double d1=(input[positpix+sizeSubImage*(line*2+2)]-input[positpix+sizeSubImage*(line*2+1)])/(2*h);
		double d2=(input[positpix+sizeSubImage*(column*2+2)]-input[positpix+sizeSubImage*(column*2+1)])/(2*h);



		if (input[positpix]>0){
			output[id]=(d1*d2)/(input[positpix]);
		}
		else{
			output[id]=100000000;
		}
		

    }

}






//n should be size(input)/2
/*extern "C"
__global__ void vec_sortRows(int n,float * value,int *index) {
	
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = 2*(idy * gridDim.x * blockDim.x + idx);
	float valtmp;
	int indtmp;
	int i;
	if (id <n*2)
    {
        
        for (i=0;i<1+(n/2);i++){//TO DO
		    if (value[id]>value[id+1]){//if yes -> change 
		        valtmp=value[id];
		        indtmp=index[id];
		        value[id]=value[id+1];
		        index[id]=index[id+1];
		        value[id+1]=valtmp;
		        index[id+1]=indtmp;
		    }
	    }

	}
}*/




extern "C"
__global__ void vec_divCorrelation (int n, float  *x,int sizeImage, float  *varImage,float  *varPSF)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
    if (id < n)
    {
		int lengthImage=sizeImage*sizeImage;
		int positInImage=id%lengthImage;
		int zposit=id/lengthImage;
		float div=(varImage[positInImage]*varPSF[zposit]);
		if (div>0){
			x[id] = x[id] / sqrt(div);
		}
        else{
			x[id] = -1;
		}
    }
}








//perform M=\sum_z(o_z*psf_z)+bckg, where * corresponds to convolution operator
extern "C"
__global__ void vec_manualFilteringTest (long n, int sizeImageX, int sizeImageY, int sizePSF,int nbPSF,float *m, float *o,  float *psf, float  *bckg,float *tmp)
{
	long idx = threadIdx.x + blockIdx.x * blockDim.x;
	long idy = threadIdx.y + blockIdx.y * blockDim.y;
	long id = idy * (long)gridDim.x * (long)blockDim.x + idx;
	
	long p_psf=(long)(id%(long)(sizePSF*sizePSF));//reste
	//long x_psf=p_psf/sizePSF;
	//long y_psf=p_psf%sizePSF;
	
	long p_image=(long)(id/(long)(sizePSF*sizePSF));//position
	long x_image=p_image/sizeImageY;
	long y_image=p_image%sizeImageY;
	
	
	if (id < n)
    {
        //initialize m to 0
		if (p_psf==0){
		    m[p_image]=0;
		}
		
        int t=0;//loop t among nb_psf
        
		
		
		//compute convolution for one single image
		tmp[id]=o[p_image+t*sizeImageX*sizeImageY]*psf[p_psf];
		//printf("0\n");
		
		////////////////////////////////////////////IMPOSSIBLE: __syncthreads does not synchronise blocks/grids
		__syncthreads();
		
		
		//printf("1\n");
		//p_image == x_image*sizeImage+y_image
		if (p_image!=x_image*sizeImageY+y_image){
		printf("ZUT %d  %d  %d\n",p_image,x_image,y_image,sizeImageY);
		}
		if (p_psf==0){
		    for (long i=0;i<sizePSF;i++){
		        for (long ii=0;ii<sizePSF;ii++){
		            //p_image==((x_image)*sizeImageY+y_image);
		            long nextY=(1+sizePSF*sizePSF)*ii+sizePSF*i;
		            long nextX=sizeImageY*sizePSF*sizePSF*i;
		            long index=p_image*sizePSF*sizePSF + nextY + nextX;
		            if (((x_image+i)<sizeImageX)&&((y_image+ii)<sizeImageY)){
		                if (index<n){
		                    if (p_image==0){
		                        //printf("%d  %d  %d  %d      x_  %d  %d   size: %d  %d  \n",i,ii,index,p_image,x_image,y_image,sizeImageY,sizePSF);
	                        }
	                        
	                        m[p_image]+=tmp[index];
                        }
                        else{
                             //printf("ZUT %d  %d  %d %d  %d\n",p_image,x_image,y_image,sizeImageY,index,n);
                        }
                    }
                    else{m[p_image]=-.1;}
                    
                }
	        }
		}
		__syncthreads();
		//for (int i=0;i<sizePSF;i++){
		/*for (int i=0;i<1;i++){
		    int power=1;

		    for (int ii=0;ii<i;ii++){

		        power*=2;

		    }

		    if ((id)%(2*power)==0){//if middle of convol kernel is even
		        int pp=power;
		        if (power

		        int index=id+power*sizePSF*sizePSF+power;

		        if (index<n){

		            //tmp[id]+=tmp[index];
		            tmp[id]=(float)index;

	            }

	        }

		}*/
			
    }



}




//convolution: apply local min
extern "C"
__global__ void vec_localMinimum(int n, int sizeImageX, int sizeImageY, int sizePSF,float *result, float *image)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	
	
	
	int x_image=id/sizeImageY;
	int y_image=id%sizeImageY;
	
	int px;
	int py;
	int i;
	int ii;
	int dist;
	if (id < n)
    {
        //initialize m to 0
        result[id]=image[id];
	    
		
		
		//search min
	    for (i=0,px=x_image-sizePSF/2;i<sizePSF;i++,px++){
	        for (ii=0,py=y_image-sizePSF/2;ii<sizePSF;ii++,py++){
	            
	            if ((px<sizeImageX)&&(py<sizeImageY)&&(px>=0)&&(py>=0)){
	                
                    dist=sqrt((float)(((sizePSF/2)-i)*((sizePSF/2)-i)+((sizePSF/2)-ii)*((sizePSF/2)-ii)));
	                if (dist<=sizePSF/2){
                        result[id]=fmin(image[px*sizeImageY+py],result[id]);
                    }
                    
                }
            }
        }
        
        
        
    }



}






//convolution: apply local max
extern "C"
__global__ void vec_localMaximum(int n, int sizeImageX, int sizeImageY, int sizePSF,float *result, float *image)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	
	
	
	int x_image=id/sizeImageY;
	int y_image=id%sizeImageY;
	
	int px;
	int py;
	int i;
	int ii;
	int dist;
	
	if (id < n)
    {
        //initialize m to 0
        result[id]=image[id];
	    
		
		
		//search max
	    for (i=0,px=x_image-sizePSF/2;i<sizePSF;i++,px++){
	        for (ii=0,py=y_image-sizePSF/2;ii<sizePSF;ii++,py++){
	            
	            if ((px<sizeImageX)&&(py<sizeImageY)&&(px>=0)&&(py>=0)){
	                dist=sqrt((float)(((sizePSF/2)-i)*((sizePSF/2)-i)+((sizePSF/2)-ii)*((sizePSF/2)-ii)));
	                if (dist<sizePSF/2){
                        result[id]=fmax(image[px*sizeImageY+py],result[id]);
                    }
                    
                }
            }
        }
        
        
        
    }



}




//convolution: apply local mean
extern "C"
__global__ void vec_localMean(int n, int sizeImageX, int sizeImageY, int sizePSF,float *result, float *image)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	
	
	
	int x_image=id/sizeImageY;
	int y_image=id%sizeImageY;
	
	int px;
	int py;
	int i;
	int ii;
	int dist;
	float mean;
	float count;
	if (id < n)
    {
        //initialize m to 0
        mean=0;
	    count=0;
		
		
		//search max
	    for (i=0,px=x_image-sizePSF/2;i<sizePSF;i++,px++){
	        for (ii=0,py=y_image-sizePSF/2;ii<sizePSF;ii++,py++){
	            
	            if ((px<sizeImageX)&&(py<sizeImageY)&&(px>=0)&&(py>=0)){
	                dist=sqrt((float)(((sizePSF/2)-i)*((sizePSF/2)-i)+((sizePSF/2)-ii)*((sizePSF/2)-ii)));
	                if (dist<=sizePSF/2){
                        mean+=image[px*sizeImageY+py];
                        count++;
                    }
                    
                }
            }
        }
        result[id]=mean/count;
        
        
    }



}




//convolution: fast if psfsize roughly < 6x6
extern "C"
__global__ void vec_manualFiltering (int n, int sizeImageX, int sizeImageY, int sizePSF,float *m, float *o,  float *psf)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int id = idy * gridDim.x * blockDim.x + idx;
	
	
	
	int x_image=id/sizeImageY;
	int y_image=id%sizeImageY;
	
	int count;
	int px;
	int py;
	int i;
	int ii;
	if (id < n)
    {
        //initialize m to 0
	    m[id]=0;
		
        
		
		
		//compute convolution for one single image
		
		
		count=0;
	    for (i=0,px=x_image-sizePSF/2;i<sizePSF;i++,px++){
	        for (ii=0,py=y_image-sizePSF/2;ii<sizePSF;ii++,py++){
	            
	            if ((px<sizeImageX)&&(py<sizeImageY)&&(px>=0)&&(py>=0)){
                    m[id]+=o[px*sizeImageY+py]*psf[i*sizePSF+ii];
                    count++;
                }
            }
        }
        m[id]/=(float)count;
    }



}







//perform M=\sum_z(o_z*psf_z)+bckg, where * corresponds to convolution operator
extern "C"
__global__ void vec_manualFilteringStacked (int n, int sizeImageX, int sizeImageY, int sizePSF,int nbPSF,float *res, float *o,  float *psf)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy = threadIdx.y + blockIdx.y * blockDim.y;
	int ids = idy * gridDim.x * blockDim.x + idx;
	
	int s=ids%nbPSF;
	int id=ids/nbPSF;
	int x_image=id/sizeImageY;
	int y_image=id%sizeImageY;
	
	int count;
	int px;
	int py;
	int i;
	int ii;
	int shift=s*sizeImageY*sizeImageX;
    
	if (id < n)
    {
        
        //initialize m to 0
        
	    res[ids]=0;
		
		//compute convolution for one single image
		
		count=0;
	    for (i=0,px=x_image-sizePSF/2;i<sizePSF;i++,px++){
	        for (ii=0,py=y_image-sizePSF/2;ii<sizePSF;ii++,py++){
	            
	            if ((px<sizeImageX)&&(py<sizeImageY)&&(px>=0)&&(py>=0)){
                    res[ids]+=o[shift+px*sizeImageY+py]*psf[i*sizePSF+ii];
                    count++;
                }
            }
        }
        res[ids]/=(float)count;
    }
}










//convolution
extern "C"
__global__ void vec_manualFilteringFast (long n, long sizeImageX, long sizeImageY, long sizePSF,float *m, float *o,  float *psf)
{
	long idx = threadIdx.x + blockIdx.x * blockDim.x;
	long idy = threadIdx.y + blockIdx.y * blockDim.y;
	long id = idy * gridDim.x * blockDim.x + idx;
	
	
	
	long p_psf=(long)(id%(long)(sizePSF*sizePSF));//reste
	//long x_psf=p_psf/sizePSF;
	//long y_psf=p_psf%sizePSF;
	
	long p_image=(long)(id/(long)(sizePSF*sizePSF));//position
	long x_image=p_image/sizeImageY;
	long y_image=p_image%sizeImageY;
	
	
	//long p_image=id/(sizePSF*sizePSF);
	//long p_psf=id%(sizePSF*sizePSF);
	
	
	//int x_image=p_image/sizeImageY;
	//int y_image=p_image%sizeImageY;
	int x_psf=(int)p_psf/sizePSF;
	int y_psf=(int)p_psf%sizePSF;
	
	
	
	if (id < n)
    {   
        
        int xx_im=x_image+x_psf-sizePSF/2;
        int yy_im=y_image+y_psf-sizePSF/2;
        long o_posit=(xx_im)*sizeImageY+(yy_im);
        //long m_posit=(p_image)*sizePSF*sizePSF+p_psf   ==   id ;
        
        if ((xx_im<sizeImageX)&&(yy_im<sizeImageY)&&(xx_im>=0)&&(yy_im>=0)){
            
            printf("m_posit:%ld   p_im:%ld   p_psf:%ld  x_psf:%d    y_psf:%d        o_posit:%ld      n:%ld    v_im:%f   v_psf:%f\n",id,p_image,p_psf,x_psf,y_psf,o_posit,n,o[o_posit],psf[p_psf]);
            
            //initialize m to 0
	        m[id]=o[o_posit]*psf[sizePSF*sizePSF - (int)p_psf - 1];
	    }
		
    }



}







//convolution
extern "C"
__global__ void vec_manualFilteringStackedFast (long n, long sizeImageX, long sizeImageY, long sizePSF, long nbPSF,float *m, float *o,  float *psf)
{
	long idx = threadIdx.x + blockIdx.x * blockDim.x;
	long idy = threadIdx.y + blockIdx.y * blockDim.y;
	long ids = idy * gridDim.x * blockDim.x + idx;
	
	long id = ids/nbPSF;
	long s = ids%nbPSF;
	
	long p_psf=(long)(id%(long)(sizePSF*sizePSF));//reste
	//long x_psf=p_psf/sizePSF;
	//long y_psf=p_psf%sizePSF;
	
	long p_image=(long)(id/(long)(sizePSF*sizePSF));//position
	long x_image=p_image/sizeImageY;
	long y_image=p_image%sizeImageY;
	
	
	//long p_image=id/(sizePSF*sizePSF);
	//long p_psf=id%(sizePSF*sizePSF);
	
	
	//int x_image=p_image/sizeImageY;
	//int y_image=p_image%sizeImageY;
	int x_psf=(int)p_psf/sizePSF;
	int y_psf=(int)p_psf%sizePSF;
	
	
	
	if (id < n)
    {   
        
        int xx_im=x_image+x_psf-sizePSF/2;
        int yy_im=y_image+y_psf-sizePSF/2;
        long o_posit=s*sizeImageX*sizeImageY + (xx_im)*sizeImageY+(yy_im);
        //long m_posit=(p_image)*sizePSF*sizePSF+p_psf   ==   id ;
        
        if ((xx_im<sizeImageX)&&(yy_im<sizeImageY)&&(xx_im>=0)&&(yy_im>=0)){
            
            //printf("m_posit:%ld   p_im:%ld   p_psf:%ld  x_psf:%d    y_psf:%d        o_posit:%ld      n:%ld    v_im:%f   v_psf:%f\n",id,p_image,p_psf,x_psf,y_psf,o_posit,n,o[o_posit],psf[p_psf]);
            
            //initialize m to 0
	        m[ids]=o[o_posit]*psf[sizePSF*sizePSF - (int)p_psf - 1];
	    }
		
    }



}









































